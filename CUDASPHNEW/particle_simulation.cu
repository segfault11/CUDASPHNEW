#include "hip/hip_runtime.h"
#include <thrust\sort.h>
#include <thrust\device_ptr.h>
#include <thrust\for_each.h>
#include <thrust\iterator\zip_iterator.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include "particle_simulation.h"
#include "util.h"
#include "cgtk\include\clock.h"
#include "boundary_map.h"
#include <thrust\scan.h>

using namespace std;

//-----------------------------------------------------------------------------
//  DEVICE CODE 
//-----------------------------------------------------------------------------

//-----------------------------------------------------------------------------
//  global device variables 
//-----------------------------------------------------------------------------
__constant__ SimulationParameters gSimParamsDev;

texture<float, hipTextureType1D, hipReadModeElementType> 
    gParticleVertexData;
texture<float, hipTextureType1D, hipReadModeElementType> 
    gParticleSimulationData;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gCellStartList;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gCellEndList;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gSortedParticleIdList;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gParticleHashList;

//  information about boundary handling
__constant__ float gBoundaryOrigin[3];
__constant__ float gDx;
__constant__ unsigned int gnBoundarySamples[3];
__constant__ float gRestDistance;

texture<float, hipTextureType1D, hipReadModeElementType> 
    gNodeTable;
texture<unsigned int, hipTextureType1D, hipReadModeElementType> 
    gIndexMap;

//-----------------------------------------------------------------------------
//  declaration of aux. functions (device) 
//-----------------------------------------------------------------------------
__device__ inline int3 compute_grid_coordinate (float3 pos, float d);
__device__ inline int compute_hash_from_grid_coordinate (int i, int j, int k);
__device__ inline float compute_distance (float3 a, float3 b);
__device__ inline float norm (const float3& a);
__device__ inline void normalize (float3& a);
__device__ inline float dot_product (const float3& a, const float3& b);
__device__ float compute_particle_density_cell (const float3 &pos, 
	float* pParticleList, int* pParticleIdList, int start, int end);
__device__ inline void compute_viscosity_pressure_forces_and_ifsurf_cell
    (const float3& xi, float rhoi, float pi, const float3& vi,
    float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int start, int end, 
    float3* force, float3* colGra, float* colLapl,
    float3* sumPosNeighbor, float* nNeighbors);

//-----------------------------------------------------------------------------
// CUDA Kernel definitions 
//-----------------------------------------------------------------------------
__global__ void compute_particle_hash (float* particleVertexData, 
    int* particleIdList, int* particleHashList, unsigned int numParticles) 
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    // calculate corresponding gridpoint
    int x = (int)((tex1Dfetch(gParticleVertexData, idx*VD_NUM_ELEMENTS + VD_POS_X) - 
		gSimParamsDev.gridOrigin[0])/gSimParamsDev.gridSpacing);
    int y = (int)((tex1Dfetch(gParticleVertexData, idx*VD_NUM_ELEMENTS + VD_POS_Y) - 
		gSimParamsDev.gridOrigin[1])/gSimParamsDev.gridSpacing);
    int z = (int)((tex1Dfetch(gParticleVertexData, idx*VD_NUM_ELEMENTS + VD_POS_Z) - 
		gSimParamsDev.gridOrigin[2])/gSimParamsDev.gridSpacing);

    // wrap outer particles to grid
    // TODO: modulo operation using "&" is faster, requires grid dims of 
	// power of two
    x = x % gSimParamsDev.gridDim[0];
    y = y % gSimParamsDev.gridDim[1];
    z = z % gSimParamsDev.gridDim[2];
    
    // calculate hash, i.e. grid cell id
    int hash = gSimParamsDev.gridDim[0]*(gSimParamsDev.gridDim[1]*z + y) + x;

    particleIdList[idx] = idx;
    particleHashList[idx] = hash;
}
//-----------------------------------------------------------------------------
__global__ void compute_sub_particle_hash (float* particleVertexData, 
    int* particleIdList, int* particleHashList, unsigned int numParticles) 
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    int id = particleIdList[idx];

    // calculate corresponding gridpoint
    int x = (int)((particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X] - 
        gSimParamsDev.gridOrigin[0])/gSimParamsDev.gridSpacingSubParticles);
    int y = (int)((particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y] - 
		gSimParamsDev.gridOrigin[1])/gSimParamsDev.gridSpacingSubParticles);
    int z = (int)((particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z] - 
		gSimParamsDev.gridOrigin[2])/gSimParamsDev.gridSpacingSubParticles);

    // wrap outer particles to grid
    // TODO: modulo operation using "&" is faster, requires grid dims of 
	// power of two
    x = x % gSimParamsDev.gridDimSubParticles[0];
    y = y % gSimParamsDev.gridDimSubParticles[1];
    z = z % gSimParamsDev.gridDimSubParticles[2];
    
    // calculate hash, i.e. grid cell id
    int hash = gSimParamsDev.gridDimSubParticles[0]*
        (gSimParamsDev.gridDimSubParticles[1]*z + y) + x;

    particleHashList[idx] = hash;
}
//-----------------------------------------------------------------------------
__global__ void compute_cell_start_end (int* particleHashList, 
	int* cellStartList,  int* cellEndList, unsigned int numParticles)
{
    extern __shared__ int sharedHash[];
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    int hash;

    if (idx < numParticles) 
    {
        hash = particleHashList[idx];
        sharedHash[threadIdx.x + 1] = hash;
        
        if (idx > 0 && threadIdx.x == 0) 
        {
            sharedHash[0] = particleHashList[idx - 1];
        }
    }

    __syncthreads();

    if (idx < numParticles) 
    {
        if (idx == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStartList[hash] = idx;
        
            if (idx > 0) 
            {
                cellEndList[sharedHash[threadIdx.x]] = idx;
            }
        }

        if (idx == numParticles - 1)
        {
            cellEndList[hash] = idx + 1;
        }
    }
}
//-----------------------------------------------------------------------------
//  Compute density and pressure for each particle 
__global__ void compute_particle_density_pressure (float* particleVertexData, 
	float* particleSimulationData, int* particleIdList, int* cellStartList, 
    int* cellEndList) 
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= gSimParamsDev.numParticles) {
        return;
    }

	int id = particleIdList[idx];

	float density = 0.0f;
    float pressure;
    float3 pos;

    // get particles position form vertex data
    pos.x = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_X);
    pos.y = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Y);
    pos.z = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Z);

    int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupport);
    int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupport);

    int hash;
    int start;
    int end;

    for(int k = c0.z; k <= c1.z; k++) 
    {
        for(int j = c0.y; j <= c1.y; j++) 
        {
            for(int i = c0.x; i <= c1.x; i++)
            {
                hash = compute_hash_from_grid_coordinate(i, j, k);
                start = tex1Dfetch(gCellStartList, hash);
                end = tex1Dfetch(gCellEndList, hash);
                density += compute_particle_density_cell(pos, 
                    particleVertexData, particleIdList, start, end);
            }
        }
    }
    
    density *= gSimParamsDev.particleMass;
    pressure = gSimParamsDev.gasStiffness*(density - 
        gSimParamsDev.restDensity);

    // set density and pressure
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_DENSITY] = density;
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_PRESSURE] = pressure; 
}
//-----------------------------------------------------------------------------
__global__ void compute_particle_acceleration_ifsurf 
    (float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int* cellStartList,
    int* cellEndList, int* isSurfaceParticle)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }
    
    int id = tex1Dfetch(gSortedParticleIdList, idx);

    float density  = tex1Dfetch(gParticleSimulationData, id*SD_NUM_ELEMENTS
        + SD_DENSITY);
    float pressure = tex1Dfetch(gParticleSimulationData, id*SD_NUM_ELEMENTS
        + SD_PRESSURE);
    float tenCoeff = gSimParamsDev.tensionCoefficient;

    float3 pos;
    pos.x = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_X);
    pos.y = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Y);
    pos.z = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Z);

    float3 vel;
    vel.x = tex1Dfetch(gParticleSimulationData, 
        id*SD_NUM_ELEMENTS + SD_VEL0_X);
    vel.y = tex1Dfetch(gParticleSimulationData, 
        id*SD_NUM_ELEMENTS + SD_VEL0_Y);
    vel.z = tex1Dfetch(gParticleSimulationData, 
        id*SD_NUM_ELEMENTS + SD_VEL0_Z);

    int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupport);
    int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupport);

    float3 force;
    force.x = 0.0f;
    force.y = 0.0f;
    force.z = 0.0f;

    float3 colGra;
    colGra.x = 0.0f;
    colGra.y = 0.0f;
    colGra.z = 0.0f;

    // [sumPosNeigbor] and [nNeigbors] are used to computed the center of mass
    // of the neighborhood of this particle (this also includes the particle
    // itself
    float3 sumPosNeighbor;
    sumPosNeighbor.x = pos.x;
    sumPosNeighbor.x = pos.y;
    sumPosNeighbor.x = pos.z;

    float nNeighbors = 1.0f;

    float colLapl;
    float colGraNorm;
    float grav = gSimParamsDev.gravity;

    int hash;
    int start;
    int end;

    // compute viscosity and pressure forces
    for(int k = c0.z; k <= c1.z; k++)
    {
        for(int j = c0.y; j <= c1.y; j++)
        {
            for(int i = c0.x; i <= c1.x; i++)
            {
                hash  = compute_hash_from_grid_coordinate(i, j, k);
                start = tex1Dfetch(gCellStartList, hash);
                end = tex1Dfetch(gCellEndList, hash);
                compute_viscosity_pressure_forces_and_ifsurf_cell(pos, density, 
                    pressure, vel, particleVertexData, particleSimulationData,
                    particleIdList, start, end, &force, &colGra, &colLapl,
                    &sumPosNeighbor, &nNeighbors);
            }
        }
    }

    // surface tension
    colGraNorm = sqrtf(colGra.x*colGra.x + colGra.y*colGra.y 
        + colGra.z*colGra.z);

    float fCoeff = tenCoeff*colLapl/colGraNorm;

    if(colGraNorm > gSimParamsDev.normThresh) 
    {
        force.x -= fCoeff*colGra.x;
        force.y -= fCoeff*colGra.y;
        force.z -= fCoeff*colGra.z;
    }



    // compute contribution of boundary to the pressure force
    /*unsigned int i, j, k;
    
    i = (unsigned int)((pos.x - gBoundaryOrigin[0])/gDx);
    j = (unsigned int)((pos.y - gBoundaryOrigin[1])/gDx);
    k = (unsigned int)((pos.z - gBoundaryOrigin[2])/gDx);

    unsigned int idx2 = i + gnBoundarySamples[0]*(j + gnBoundarySamples[1]*k);
    unsigned int nodeIdx = tex1Dfetch(gIndexMap, idx2);
    float dist = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_DISTANCE);
    
    float3 bNorm;

    bNorm.x = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_X);
    bNorm.y = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_Y);
    bNorm.z = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_Z);
    
    float3 boundaryForce;
    float bCoeff;

    bCoeff = gSimParamsDev.particleMass*(gRestDistance - dist)/
        (gSimParamsDev.timeStep*gSimParamsDev.timeStep);

    boundaryForce.x = bCoeff*bNorm.x;
    boundaryForce.y = bCoeff*bNorm.y;
    boundaryForce.z = bCoeff*bNorm.z;*/

    
    // store the actual acceleration
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_X] = force.x/density;  
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_Y] = force.y/density
        - grav;  
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_Z] = force.z/density;  

    // find out if particle is a surface particle
    /*int isSurface = 0;

    float3 dCenterMass;
    dCenterMass.x = pos.x - sumPosNeighbor.x/nNeighbors;
    dCenterMass.y = pos.y - sumPosNeighbor.y/nNeighbors;
    dCenterMass.z = pos.z - sumPosNeighbor.z/nNeighbors;

    float dCenterMassNormSq = dCenterMass.x*dCenterMass.x +
        dCenterMass.y*dCenterMass.y + dCenterMass.z*dCenterMass.z;

    if (colGraNorm > gSimParamsDev.normThresh) {
        isSurfaceParticle[id] = 1;
    } else {
        isSurfaceParticle[id] = 0;
    }*/
}
//----------------------------------------------------------------------------
/*__global__ void compute_sub_particle_acceleration
    (float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int* cellStartList,
    int* cellEndList, int* isSurfaceParticle)
{

}*/
//-----------------------------------------------------------------------------
__global__ void integrate_euler (float* particleVertexData, 
    float* particleSimulationData)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int idVert = idx*VD_NUM_ELEMENTS;
    unsigned int idSim = idx*SD_NUM_ELEMENTS;
    float dt = gSimParamsDev.timeStep;

    particleSimulationData[idSim + SD_VEL0_X] += 
        dt*particleSimulationData[idSim + SD_ACC_X];
    particleSimulationData[idSim + SD_VEL0_Y] += 
        dt*particleSimulationData[idSim + SD_ACC_Y];
    particleSimulationData[idSim + SD_VEL0_Z] += 
        dt*particleSimulationData[idSim + SD_ACC_Z];

    particleVertexData[idVert + VD_POS_X] += 
        dt*particleSimulationData[idSim + SD_VEL0_X];
    particleVertexData[idVert + VD_POS_Y] += 
        dt*particleSimulationData[idSim + SD_VEL0_Y];
    particleVertexData[idVert + VD_POS_Z] += 
        dt*particleSimulationData[idSim + SD_VEL0_Z];   
}
//-----------------------------------------------------------------------------
__global__ void integrate_sub_particles_euler (float* subParticleVertexData, 
    float* subParticleSimulationData, int* subParticleIds, 
    unsigned int nSubParticles)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= nSubParticles)
    {
        return;
    }

    int id = subParticleIds[idx];
    unsigned int idVert = id*VD_NUM_ELEMENTS;
    unsigned int idSim = id*SD_NUM_ELEMENTS;
    float dt = gSimParamsDev.timeStep;
    
    
    subParticleSimulationData[idSim + SD_VEL0_X] += 
        dt*subParticleSimulationData[idSim + SD_ACC_X];
    subParticleSimulationData[idSim + SD_VEL0_Y] += 
        dt*subParticleSimulationData[idSim + SD_ACC_Y];
    subParticleSimulationData[idSim + SD_VEL0_Z] += 
        dt*subParticleSimulationData[idSim + SD_ACC_Z];
    
    subParticleVertexData[idVert + VD_POS_X] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_X];
    subParticleVertexData[idVert + VD_POS_Y] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_Y];
    subParticleVertexData[idVert + VD_POS_Z] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_Z];
}
//-----------------------------------------------------------------------------
__global__ void shift_state (char* particleState)
{
        unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

        if (idx >= gSimParamsDev.numParticles)
        {
            return;        
        }

        particleState[idx] = (particleState[idx] << 2);
}
//-----------------------------------------------------------------------------
__global__ void collision_handling (float* particleVertexData, 
    float* particleSimulationData)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int idVert = idx*VD_NUM_ELEMENTS;
    unsigned int idSim = idx*SD_NUM_ELEMENTS;

    float3 pos;
    float3 vel;

    pos.x = tex1Dfetch(gParticleVertexData, idVert + VD_POS_X);
    pos.y = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Y);
    pos.z = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Z);

    vel.x = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_X);
    vel.y = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Y);
    vel.z = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Z);

    float3 local;
    float3 diff;
    float3 nrm;

    float dist;
    float depth;

    // compute "distance" to box, if positive the particle
    // is outside the box.

    // compute local position of the particle to the box
    local.x = pos.x - gSimParamsDev.boxCen[0];
    local.y = pos.y - gSimParamsDev.boxCen[1];
    local.z = pos.z - gSimParamsDev.boxCen[2];

    // project local pos to the upper right quadrand and
    // compute difference to the boxDim vec
    diff.x = abs(local.x) - gSimParamsDev.boxDim[0];
    diff.y = abs(local.y) - gSimParamsDev.boxDim[1];
    diff.z = abs(local.z) - gSimParamsDev.boxDim[2];

    dist = max(diff.x, diff.y);
    dist = max(dist, diff.z);
    
    // if the particle lies outside the box, the collision must be handled
    float3 contact;
    
    if (dist > 0.0f)
    {
        // contact point in "box space"
        contact.x = min(gSimParamsDev.boxDim[0], 
            max(-gSimParamsDev.boxDim[0], local.x));
        contact.y = min(gSimParamsDev.boxDim[1],
            max(-gSimParamsDev.boxDim[1], local.y));
        contact.z = min(gSimParamsDev.boxDim[2],
            max(-gSimParamsDev.boxDim[2], local.z));

        // translate to worldspace
        contact.x += gSimParamsDev.boxCen[0];
        contact.y += gSimParamsDev.boxCen[1];
        contact.z += gSimParamsDev.boxCen[2];

        // compute penetration depth
        depth = compute_distance(contact, pos);

        // compute normal
        nrm.x = pos.x - contact.x;
        nrm.y = pos.y - contact.y;
        nrm.z = pos.z - contact.z;
        normalize(nrm);

        float velNorm = norm(vel);
        float dp     = dot_product(nrm, vel);
        float coeff  = (1 + gSimParamsDev.restitution*depth/
            (gSimParamsDev.timeStep*velNorm))*dp;

        vel.x -= coeff*nrm.x;
        vel.y -= coeff*nrm.y;
        vel.z -= coeff*nrm.z;

        particleVertexData[idVert + VD_POS_X] = contact.x;
        particleVertexData[idVert + VD_POS_Y] = contact.y;
        particleVertexData[idVert + VD_POS_Z] = contact.z;

        particleSimulationData[idSim + SD_VEL0_X] = vel.x;
        particleSimulationData[idSim + SD_VEL0_Y] = vel.y;
        particleSimulationData[idSim + SD_VEL0_Z] = vel.z;
    }
}
//-----------------------------------------------------------------------------
__global__ void update_particle_state (float* particleVertexData, 
    char* particleState, int* particleIdList, int* cellStartList, 
    int* cellEndList)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int id = particleIdList[idx];
    float3 pos;
    float3 xj;
    float3 r;
    float rn;
    pos.x = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];

    if (pos.x >= 0.2f && pos.x <= 0.5f)
    {
        particleState[id] |= 1;

        // distribute information to neigbors
        pos.y = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
        pos.z = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

        int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupport);
        int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupport);
        
        int hash;
        int start;
        int end;
        
        for(int k = c0.z; k <= c1.z; k++)
        {
            for(int j = c0.y; j <= c1.y; j++)
            {
                for(int i = c0.x; i <= c1.x; i++)
                {
                    hash  = compute_hash_from_grid_coordinate(i, j, k);
                    start = cellStartList[hash];
                    end = cellEndList[hash];
                    
                    for (int u = start; u < end; u++) 
                    {
                        int v = particleIdList[u]; 
                        xj.x = particleVertexData[v*VD_NUM_ELEMENTS + VD_POS_X];
                        xj.y = particleVertexData[v*VD_NUM_ELEMENTS + VD_POS_Y];                         
                        xj.z = particleVertexData[v*VD_NUM_ELEMENTS + VD_POS_Z];
                        r.x = pos.x - xj.x;
                        r.y = pos.y - xj.y;
                        r.z = pos.z - xj.z;
                        rn = r.x*r.x + r.y*r.y + r.z*r.z;

                        if (rn <= gSimParamsDev.compactSupport*
                            gSimParamsDev.compactSupport)
                        {
                            particleState[v] |= 2;
                        }
                    }
                }
            }
        }
        
        // why is this neccessary?
        particleState[id] |= 1;
    }
}
//-----------------------------------------------------------------------------
// sets the sub particle vertex & simulation data for each particle, that has
// changed its state from "default" to "boundary" or "split".
// This kernel is called for particles that were split and for boundary 
// particles. [numParticles] refers to the total number of those particles.
__global__ void initialize_sub_particles (float* subParticleVertexData, 
    float* subParticleSimulationData, int* particleIds, 
    float* particleVertexData, float* particleSimulationData, 
    char* particleStates, unsigned int numParticles)
{
#define SQRT3INV 0.577350269 // = 1/sqrt{3}

    // directions to seed the new sub particles
    const float directions[] = {
                                    SQRT3INV,  SQRT3INV,  SQRT3INV, 
                                    SQRT3INV,  SQRT3INV, -SQRT3INV,
                                    SQRT3INV, -SQRT3INV,  SQRT3INV,
                                    SQRT3INV, -SQRT3INV, -SQRT3INV,
                                   -SQRT3INV,  SQRT3INV,  SQRT3INV,
                                   -SQRT3INV,  SQRT3INV, -SQRT3INV,
                                   -SQRT3INV, -SQRT3INV,  SQRT3INV,
                                   -SQRT3INV, -SQRT3INV, -SQRT3INV
                               };

    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    unsigned int id = particleIds[idx];
    char state = particleStates[id];

    // if the base particle was "default" previously ...
    if ((state & 12) == 0)
    {
        float density = particleSimulationData[id*SD_NUM_ELEMENTS + SD_DENSITY];
        float radicand = 3.0f*gSimParamsDev.particleMass/(4.0f*M_PI*density);
        float radius = pow(radicand, 1.0f/3.0f);
        float3 pos;
        pos.x = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];
        pos.y = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
        pos.z = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

        // ... initialize initial position and velocity of the corresponding 
        // sub particles
        for (unsigned int i = 0; i < 8; i++) 
        {
            // update velocity
            int index = (8*id + i)*SD_NUM_ELEMENTS;
            subParticleSimulationData[index + SD_VEL0_X] = 0.0f;
                particleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_X];
            subParticleSimulationData[index + SD_VEL0_Y] = 0.1f;
                particleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_Y];
            subParticleSimulationData[index + SD_VEL0_Z] = 0.0f;
                particleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_Z];
           
            // update position
            index = (8*id + i)*VD_NUM_ELEMENTS;
            subParticleVertexData[index + VD_POS_X] = pos.x + 
                directions[3*i + 0]*radius;
            subParticleVertexData[index + VD_POS_Y] = pos.y + 
                directions[3*i + 1]*radius;
            subParticleVertexData[index + VD_POS_Z] = pos.z + 
                directions[3*i + 2]*radius;
        }
    }

#undef SQRT3INV
}
//-----------------------------------------------------------------------------
__global__ void check_split_boundary_default (char* particleState, 
        int* isSplit, int* isBoundary, int* isDefault)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    char state = particleState[idx] & 3;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    if (state == 0)
    {
        isDefault[idx] = 1;
    }
    else if (state == 2)
    {
        isBoundary[idx] = 1;
    }
    else
    {
        isSplit[idx] = 1;
    }
}
//-----------------------------------------------------------------------------
__global__ void collect_ids (int* subParticleIdList, int* splitParticleIdList,
    int* boundaryParticleIdList, int* defaultParticleIdList,
    int* isSplit, int* isBoundary, int* isDefault, int* splitPrefixSum, 
    int* boundaryPrefixSum, int* defaultPrefixSum, 
    unsigned int numParticlesSplit)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    if (isSplit[idx] == 1)
    {
        int splitPreSum = splitPrefixSum[idx];

        for (unsigned int i = 0; i < 8; i++)
        {
            subParticleIdList[8*splitPreSum + i] = 8*idx + i;
        }

        splitParticleIdList[splitPreSum] = idx; 
    }
    else if (isBoundary[idx] == 1)
    {
        int boundaryPreSum = boundaryPrefixSum[idx];

        for (unsigned int i = 0; i < 8; i++)
        {
            subParticleIdList[8*(numParticlesSplit + boundaryPreSum) + i] =  
                8*idx + i;
        }

        boundaryParticleIdList[boundaryPreSum] = idx;
    }
    else
    {
        defaultParticleIdList[defaultPrefixSum[idx]] = idx;
    }
}
//__global__ void collision_handling(float* particleVertexData, 
//    float* particleSimulationData)
//{
//    /*unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
//
//    if (idx >= gSimParamsDev.numParticles)
//    {
//        return;
//    }
//
//    unsigned int idVert = idx*VD_NUM_ELEMENTS;
//    unsigned int idSim = idx*SD_NUM_ELEMENTS;
//
//    float3 pos;
//    float3 vel;
//
//    pos.x = tex1Dfetch(gParticleVertexData, idVert + VD_POS_X);
//    pos.y = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Y);
//    pos.z = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Z);
//
//    vel.x = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_X);
//    vel.y = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Y);
//    vel.z = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Z);
//
//    //
//    unsigned int i,j,k;
//    i = (unsigned int)((pos.x - gBoundaryOrigin[0])/gDx);
//    j = (unsigned int)((pos.y - gBoundaryOrigin[1])/gDx);
//    k = (unsigned int)((pos.z - gBoundaryOrigin[2])/gDx);
//    unsigned int idx2 = i + gnBoundarySamples[0]*(j + gnBoundarySamples[1]*k);
//    unsigned int nodeIdx = tex1Dfetch(gIndexMap, idx2);
//    float dist = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_DISTANCE);
//    
//    float3 bNorm;
//
//    bNorm.x = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_X);
//    bNorm.y = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_Y);
//    bNorm.z = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_Z);
//
//    if (bNorm.y != 0.0f)
//    {
//        particleVertexData[idVert + VD_POS_X] -= gSimParamsDev.timeStep*vel.x;
//        particleVertexData[idVert + VD_POS_Y] -= gSimParamsDev.timeStep*vel.y;
//        particleVertexData[idVert + VD_POS_Z] -= gSimParamsDev.timeStep*vel.z;
//    }*/
//}

//-----------------------------------------------------------------------------
// definition of aux. functions (device) 
//-----------------------------------------------------------------------------
__device__ inline int3 compute_grid_coordinate(float3 pos, float d)
{
    int3 gridCoord;

    gridCoord.x = (unsigned int)((pos.x + d - gSimParamsDev.gridOrigin[0])/
        gSimParamsDev.gridSpacing);
    gridCoord.y = (unsigned int)((pos.y + d - gSimParamsDev.gridOrigin[1])/
        gSimParamsDev.gridSpacing);
    gridCoord.z = (unsigned int)((pos.z + d - gSimParamsDev.gridOrigin[2])/
        gSimParamsDev.gridSpacing);

    gridCoord.x = gridCoord.x%gSimParamsDev.gridDim[0];
    gridCoord.y = gridCoord.y%gSimParamsDev.gridDim[1];
    gridCoord.z = gridCoord.z%gSimParamsDev.gridDim[2];

    gridCoord.x = min(max(gridCoord.x, 0),gSimParamsDev.gridDim[0] - 1);
    gridCoord.y = min(max(gridCoord.y, 0),gSimParamsDev.gridDim[1] - 1);
    gridCoord.z = min(max(gridCoord.z, 0),gSimParamsDev.gridDim[2] - 1);

    return gridCoord;
}
//-----------------------------------------------------------------------------
__device__ inline int compute_hash_from_grid_coordinate(int i, int j, int k)
{
    return gSimParamsDev.gridDim[0]*(gSimParamsDev.gridDim[1]*k + j) + i;
}
//-----------------------------------------------------------------------------
__device__ inline float norm(const float3& a)
{
    return sqrt(a.x*a.x+a.y*a.y+a.z*a.z);
}
//-----------------------------------------------------------------------------
__device__ inline void normalize (float3& a)
{
    float norm = sqrt(a.x*a.x+a.y*a.y+a.z*a.z);
    a.x /= norm;
    a.y /= norm;
    a.z /= norm;
}
//-----------------------------------------------------------------------------
//  Computes the Euclidean distance between two points.
__device__ inline float compute_distance (float3 a, float3 b)
{
    return sqrt((a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y) 
        + (a.z-b.z)*(a.z-b.z));
}
//-----------------------------------------------------------------------------
__device__ inline float dot_product (const float3& a, const float3& b)  
{
    return a.x*b.x + a.y*b.y + a.z*b.z;
}
//-----------------------------------------------------------------------------
//  Computes the contribution of neighborparticles of one particular grid cell
//  to the density of the particle at position [pos].
__device__ float compute_particle_density_cell (const float3 &pos, 
	float* particleVertexData, int* particleIdList, int start, int end)
{
    int particleIndex; // index of the neighbor of the particle
    float density = 0.0f;
    float3 p; // neighbor particle's position
    float h = gSimParamsDev.compactSupport;
    float r;
    float d;

    for (int i = start; i < end; i++) 
    {
        particleIndex = tex1Dfetch(gSortedParticleIdList, i);

        // compute position of the neighbor
        p.x = tex1Dfetch(gParticleVertexData, particleIndex*VD_NUM_ELEMENTS
            + VD_POS_X);
        p.y = tex1Dfetch(gParticleVertexData, particleIndex*VD_NUM_ELEMENTS
            + VD_POS_Y);
        p.z = tex1Dfetch(gParticleVertexData, particleIndex*VD_NUM_ELEMENTS
            + VD_POS_Z);

        r = compute_distance(p, pos);
        
        // TODO: evaluating r*r <= h*h might save taking the sqrt in 
        // compute_distance proc. 
        if (r <= h) 
        {
            d = h*h - r*r;
            density += gSimParamsDev.poly6*d*d*d;
        }
    }

    return density;
}
//-----------------------------------------------------------------------------
__device__ inline void compute_viscosity_pressure_forces_and_ifsurf_cell
    (const float3& xi, float rhoi, float pi, const float3& vi,
    float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int start, int end, 
    float3* force, float3* colGra, float* colLapl,
    float3* sumPosNeighbor, float* nNeighbors)
{
    int j;      // neighbor index in particle list
    float3 xj;  // neighbor particle's position
    float3 vj;  // neighbor particle's velocity
    float rhoj; // neighbor density
    float pj;   // neighbor pressure
    float3 r;   // xi - xj
    float rn;   // ||xi - xj||
    float h = gSimParamsDev.compactSupport; // effective radius
    float grad  = gSimParamsDev.gradSpiky;
    float lapl  = gSimParamsDev.laplVisc;
    float grad2 = gSimParamsDev.gradPoly6;
    float lapl2 = gSimParamsDev.laplPoly6;

    float pressure; // pressure term in the kernel approx
    float rhoi2 = rhoi*rhoi;                    
    float m = gSimParamsDev.particleMass;
    float mu = gSimParamsDev.dynamicViscosity;

    float d; // helper value to avoid arithmetic operations

    for (int i = start; i < end; i++) 
    {
        // get neighbor index from particle list
        j = tex1Dfetch(gSortedParticleIdList, i); 

        // get neighbor particle information
        xj.x = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS + VD_POS_X);
        xj.y = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS + VD_POS_Y);
        xj.z = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS + VD_POS_Z);
        vj.x = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_VEL0_X);
        vj.y = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_VEL0_Y);
        vj.z = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_VEL0_Z);
        rhoj = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_DENSITY);
        pj   = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_PRESSURE);

        r.x = xi.x - xj.x;
        r.y = xi.y - xj.y;
        r.z = xi.z - xj.z;

        rn = norm(r);
        
        // TODO: * masse koennte ausgeklammert werden um multiplikationen
        //         zu sparen.
        //       * generell kann der pressure term in hinblick auf rhoi und
        //         pi vereinfacht werden.
        //       * visc force: mu koennte ausgeklammert werden etc.
        //       * zwei float3's fuer beide kraefte koennten genutzt werden
        //         um die terme zu vereinfachen.
        pressure = rhoi*(pi/rhoi2 + pj/(rhoj*rhoj))*m;

        if (rn <= h && rn > 0.0f)
        {
            // compute pressure force
            d = (h-rn)*(h-rn);

            force->x -= pressure*grad*d/rn*r.x;
            force->y -= pressure*grad*d/rn*r.y;
            force->z -= pressure*grad*d/rn*r.z;
        
            // compute viscosity force
            d = (h-rn);

            force->x += mu*(vj.x-vi.x)*m/rhoj*lapl*d;
            force->y += mu*(vj.y-vi.y)*m/rhoj*lapl*d;
            force->z += mu*(vj.z-vi.z)*m/rhoj*lapl*d;

            // compute color gradient
            d = (h*h-rn*rn)*(h*h-rn*rn);

            colGra->x += m/rhoj*grad2*d*r.x;
            colGra->y += m/rhoj*grad2*d*r.y;
            colGra->z += m/rhoj*grad2*d*r.z;

            // compute color laplacian
            d = (h*h - rn*rn)*(3.0f*h*h - 7.0f*rn*rn);

            *colLapl += m/rhoj*lapl2*d;

            //
            sumPosNeighbor->x += xj.x;
            sumPosNeighbor->y += xj.y;
            sumPosNeighbor->z += xj.z;
            *nNeighbors += 1.0f;
        }
    }
}

//-----------------------------------------------------------------------------
//  HOST CODE
//-----------------------------------------------------------------------------

#define EMPTY_CELL 0xFFFFFFFF

//-----------------------------------------------------------------------------
//  forward declaration of aux. functions
//-----------------------------------------------------------------------------
void create_particle_box (float sx, float sy, float sz, float d, 
    unsigned int numParticles, float** particleVD, float** particleSD,
    unsigned int* numParticlesCreated);
void set_simulation_domain (float xs, float ys, float zs, float xe,
    float ye, float ze, float gridSpacing, float gridSpacingSubParticles,
    SimulationParameters* parameters);
void compute_particle_kernel_invocation_information 
    (unsigned int& nThreadsBlock, unsigned int& nBlocks, 
    unsigned int numParticles);


//-----------------------------------------------------------------------------
//  Definition of ParticleSimulation class 
//-----------------------------------------------------------------------------
/* Set everything to NULL/0
*/
ParticleSimulation::ParticleSimulation (): mParticleVertexData(NULL), 
    mParticleSimulationData(NULL), mParticleVertexDataDevPtr(NULL),
    mParticleSimulationDataDevPtr(NULL), mParticleIdsDevPtr(NULL),
    mParticleHashListDevPtr(NULL), mCellStartListDevPtr(NULL), 
    mCellEndListDevPtr(NULL), mIsSurfaceParticleDevPtr(NULL), mParticleVertexDataVbo(0),
    mNumBlocks(0), mThreadsPerBlock(0), mNumSubParticles(0)
{
    memset(&mParameters, 0, sizeof(SimulationParameters));
}
//-----------------------------------------------------------------------------
ParticleSimulation::~ParticleSimulation() 
{
    this->freeAll();
}
//-----------------------------------------------------------------------------
ParticleSimulation* ParticleSimulation::Example01 () 
{
    // create a particle simulation 
    ParticleSimulation* sim = new ParticleSimulation();

    // create box (cube) of particles
    create_particle_box(-0.65f, -0.45f, -0.25f, 0.5f, 40000, 
        &sim->mParticleVertexData, &sim->mParticleSimulationData,
        &sim->mParameters.numParticles);

    if (sim->mParticleVertexData == NULL || 
        sim->mParticleSimulationData == NULL) 
    {
        THROW_EXCEPTION("Could not allocate memory for particles (Host).");
    }

    // set sph simulation related parameters
    sim->mParameters.kernelParticles = 20;
    sim->mParameters.restDensity = 998.648f;
    sim->mParameters.particleMass = sim->mParameters.restDensity*0.5f*0.5f*0.5f/
        static_cast<float>(sim->mParameters.numParticles);
    sim->mParameters.gasStiffness = 3.0f;
    sim->mParameters.dynamicViscosity = 3.0f;
    sim->mParameters.gravity = 9.81f;
    sim->mParameters.tensionCoefficient = 0.0728f;
    sim->mParameters.normThresh = 15.065f;

    // compute the kernel radius
    float h = powf((3.0f*0.5f*0.5f*0.5f*sim->mParameters.kernelParticles)/
        (4.0f*M_PI*sim->mParameters.numParticles), 1.0f/3.0f);

    sim->mParameters.compactSupport =  h;
    sim->mParameters.poly6 =  315.0f/(64.0f*M_PI*h*h*h*h*h*h*h*h*h);
    sim->mParameters.gradPoly6 = -945.0f/(32.0f*M_PI*h*h*h*h*h*h*h*h*h);
    sim->mParameters.laplPoly6 = -945.0f/(32.0f*M_PI*h*h*h*h*h*h*h*h*h);
    sim->mParameters.gradSpiky = -45.0f/(M_PI*h*h*h*h*h*h);
    sim->mParameters.laplVisc  =  45.0f/(M_PI*h*h*h*h*h*h);
    sim->mParameters.timeStep  = 0.003;
    
    set_simulation_domain(-2.5, -2.5, -2.5, 2.5, 2.5, 2.5, h, h/2.0f,
        &sim->mParameters);

    // set fluid volume
    sim->mParameters.fluidVolume = 0.5f*0.5f*0.5f; 

    // set parameters for boundary handling
    sim->mParameters.restitution = 0.0f;
    sim->mParameters.boxCen[0] = 0.0f;
    sim->mParameters.boxCen[1] = 0.0f;
    sim->mParameters.boxCen[2] = 0.0f;
    sim->mParameters.boxDim[0] = 0.7f;    
    sim->mParameters.boxDim[1] = 0.5f;    
    sim->mParameters.boxDim[2] = 0.3f;    

    // set parameters for new boundary handling
    sim->_boundaryMapFileName = std::string("icosphere.txt");

    // set parameters for surface extraction
    sim->mParameters.cmDistanceThresh = 0.5f;
    sim->mParameters.nPartTresh = 20.0f;
    sim->_leftI = 0.0f;
    sim->_rightI = 1.0f;
    //printf("h %")
    return sim;
}
//-----------------------------------------------------------------------------
int* ParticleSimulation::CreateIsParticleSurfaceList
    (const ParticleSimulation* sim)
{
    int* isSurfaceParticleList = new int[sim->mParameters.numParticles];
    
    CUDA_SAFE_CALL( hipMemcpy(isSurfaceParticleList, 
        sim->mIsSurfaceParticleDevPtr,
        sizeof(int)*sim->mParameters.numParticles, 
        hipMemcpyDeviceToHost) );

    int extr = 0;
    for (unsigned int i = 0; i < sim->mParameters.numParticles; i++) {
       extr += isSurfaceParticleList[i];
    }

    printf("%d of %d extracted\n", extr, sim->mParameters.numParticles);
    

    return isSurfaceParticleList;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::FreeIsParticleSurfaceList 
    (int** isSurfaceParticleList)
{
    if (*isSurfaceParticleList == NULL) 
    {
        return;
    }

    delete[] *isSurfaceParticleList;
    *isSurfaceParticleList = NULL;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::freeAll () 
{
    // free host memory
    saveDeleteArray<float>(&mParticleVertexData);
    saveDeleteArray<float>(&mParticleSimulationData);
    
    // free device memory

    // free cuda memory
    cudaSafeFree<float>(&mParticleSimulationDataDevPtr);
    cudaSafeFree<int>(&mParticleIdsDevPtr);
    cudaSafeFree<int>(&mParticleHashListDevPtr);
    cudaSafeFree<int>(&mCellStartListDevPtr);
    cudaSafeFree<int>(&mCellEndListDevPtr);
    cudaSafeFree<int>(&mIsSurfaceParticleDevPtr);
    
    // free OpenGL vertex buffer object
    if (mParticleVertexDataVbo != 0) 
    {
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[0]) );
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[1]) );
        //cudaGLUnregisterBufferObject(mParticleVertexDataVbo); // <- deprecated
        glDeleteBuffers(1, &mParticleVertexDataVbo);
        glDeleteBuffers(1, &mSubParticleVertexDataVbo);
        mParticleVertexDataVbo = 0;
        mSubParticleVertexDataVbo = 0;
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::Init () 
{
    //
    // free device memory, if previously allocated 
    //

    // free cuda memory
    cudaSafeFree<float>(&mParticleSimulationDataDevPtr);
    cudaSafeFree<int>(&mParticleIdsDevPtr);
    cudaSafeFree<int>(&mParticleHashListDevPtr);
    cudaSafeFree<int>(&mCellStartListDevPtr);
    cudaSafeFree<int>(&mCellEndListDevPtr);
    cudaSafeFree<int>(&mIsSurfaceParticleDevPtr);
    
    // free OpenGL vertex buffer object
    if (mParticleVertexDataVbo != 0) 
    {
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[0]) );
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[1]) );
        glDeleteBuffers(1, &mParticleVertexDataVbo);
        glDeleteBuffers(1, &mSubParticleVertexDataVbo);
        mParticleVertexDataVbo = 0;
        mSubParticleVertexDataVbo = 0;
    }

    //
    // allocate cuda device memory for storing the particles' vertex and
    // simulation data.
    // Vertex data is allocated on device using OpenGL, as it is stored
    // in an vertex buffer object, which is used for rendering later.
    //

    // Simulation data is allocated through cuda.
    CUDA_SAFE_CALL( hipMalloc(&mParticleSimulationDataDevPtr, 
        mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS) );

    // copy initial host data to device
    CUDA_SAFE_CALL( hipMemcpy(mParticleSimulationDataDevPtr, 
        mParticleSimulationData, 
        mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS,
        hipMemcpyHostToDevice) );
    
    // Vertex data is allocated through a vertex buffer object
    // the vbo is then registered to be used with CUDA
    glGenBuffers(1, &mParticleVertexDataVbo);
    glBindBuffer(GL_ARRAY_BUFFER, mParticleVertexDataVbo);
    glBufferData(GL_ARRAY_BUFFER, 
        mParameters.numParticles*VD_NUM_ELEMENTS*sizeof(float),
          mParticleVertexData, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[0], 
        mParticleVertexDataVbo, cudaGraphicsMapFlagsNone) );
    //cudaGLRegisterBufferObject(mParticleVertexDataVbo); // <- is deprecated
    
    //
    // alloc & Init additional aux. arrays for nearest neighbor search
    //
    const int* dim = mParameters.gridDim; 
    unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(int);

    CUDA_SAFE_CALL( hipMalloc(&mCellStartListDevPtr, size) );
    CUDA_SAFE_CALL( hipMalloc(&mCellEndListDevPtr, size) );

    // set each cell to be empty
    CUDA_SAFE_CALL( hipMemset(mCellStartListDevPtr, EMPTY_CELL, size) );
    CUDA_SAFE_CALL( hipMemset(mCellEndListDevPtr, EMPTY_CELL, size) );
     
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleHashListDevPtr, 
        mParameters.numParticles*sizeof(int)) );

    // alloc dev memory for surface particle extraction
    CUDA_SAFE_CALL( hipMalloc(&mIsSurfaceParticleDevPtr, 
        mParameters.numParticles*sizeof(int)) );

    this->allocateMemoryTwoScale();

    // set up textures, for faster memory look-ups through caching
    // NOTE: VertexData needs to be mapped to get a valid device pointer, 
    //       as it is initial not allocated through CUDA's malloc
    hipChannelFormatDesc descf = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat);
    hipChannelFormatDesc desci = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindSigned);
    hipChannelFormatDesc descu = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindUnsigned);

    CUDA_SAFE_CALL ( hipBindTexture(0, gParticleSimulationData, 
        mParticleSimulationDataDevPtr, descf, 
        sizeof(float)*SD_NUM_ELEMENTS*mParameters.numParticles) );
    this->map();
    CUDA_SAFE_CALL ( hipBindTexture(0, gParticleVertexData, 
        mParticleVertexDataDevPtr, descf, 
        sizeof(float)*VD_NUM_ELEMENTS*mParameters.numParticles) );
    this->unmap();
    CUDA_SAFE_CALL ( hipBindTexture(0, gCellStartList, mCellStartListDevPtr, 
        desci, size) );
    CUDA_SAFE_CALL ( hipBindTexture(0, gCellEndList, mCellEndListDevPtr, 
        desci, size) );
    CUDA_SAFE_CALL ( hipBindTexture(0, gSortedParticleIdList, mParticleIdsDevPtr, 
        desci, mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL ( hipBindTexture(0, gParticleHashList, mParticleHashListDevPtr, 
        desci, mParameters.numParticles*sizeof(int)) );

    // set number of CUDA blocks and threads per blocks for each kernel 
    // invocation
    // NOTE:  - chose different values than 256 to try to get more performance
    //        - make threadsPerBlock and blocks function parameters
    compute_particle_kernel_invocation_information(mThreadsPerBlock, mNumBlocks, 
        mParameters.numParticles);
    //mThreadsPerBlock = mParameters.numParticles < 256 ? 
    //    mParameters.numParticles : 256;
    //mNumBlocks = mParameters.numParticles % mThreadsPerBlock == 0 ?
    //    mParameters.numParticles/mThreadsPerBlock : 
    //    mParameters.numParticles/mThreadsPerBlock + 1; 

    
    //
    // Init boundary handling
    //
    /*std::cout << "loading boundary information ... " << std::endl;
    BoundaryMap bmap("icosphere.txt");
    std::cout << "finished loading" << std::endl;
   
    unsigned int nCoords = bmap.GetNumCoordinates();
    unsigned int totalSamples = bmap.GetNumTotalSamples();

    CUDA_SAFE_CALL( hipMalloc(&_boundaryMapIndexMapDevPtr, 
        totalSamples*sizeof(unsigned int)) );
    
    CUDA_SAFE_CALL( hipMalloc(&_boundaryMapNodeTableDevPtr, 
        NC_NUM_ELEMENTS*nCoords*sizeof(float)) );


    CUDA_SAFE_CALL( hipMemcpy(_boundaryMapIndexMapDevPtr, bmap.GetIndexMap(),
        sizeof(unsigned int)*totalSamples, hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL( hipMemcpy(_boundaryMapNodeTableDevPtr, bmap.GetNodeTable(),
        NC_NUM_ELEMENTS*nCoords*sizeof(float), hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL ( hipBindTexture(0, gIndexMap, _boundaryMapIndexMapDevPtr, 
        descu, totalSamples*sizeof(unsigned int)) );

    CUDA_SAFE_CALL ( hipBindTexture(0, gNodeTable, _boundaryMapNodeTableDevPtr, 
        descf, NC_NUM_ELEMENTS*nCoords*sizeof(float)) );

    unsigned int nSamples[3];
    nSamples[0] = bmap.GetIMax();
    nSamples[1] = bmap.GetJMax();
    nSamples[2] = bmap.GetKMax();

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gnBoundarySamples), nSamples, 
		3*sizeof(unsigned int), 0, hipMemcpyHostToDevice) );

    float origin[3];
    origin[0] = bmap.GetDomain().getV1().getX();
    origin[1] = bmap.GetDomain().getV1().getY();
    origin[2] = bmap.GetDomain().getV1().getZ();

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gBoundaryOrigin), origin, 
		3*sizeof(float), 0, hipMemcpyHostToDevice) );

    float dx = bmap.GetDx();

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gDx), &dx, 
		sizeof(float), 0, hipMemcpyHostToDevice) );

    float restDist = bmap.GetRestDistance();

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gRestDistance), &restDist, 
		sizeof(float), 0, hipMemcpyHostToDevice) );*/

}
//-----------------------------------------------------------------------------
// allocates and initializes memory needed for the two scale particle 
// simulation
void ParticleSimulation::allocateMemoryTwoScale ()
{
    // create opengl vbo for storing the vertex information of the 
    // sub particles
    glGenBuffers(1, &mSubParticleVertexDataVbo);
    glBindBuffer(GL_ARRAY_BUFFER, mSubParticleVertexDataVbo);
    glBufferData(GL_ARRAY_BUFFER, 8*mParameters.numParticles*VD_NUM_ELEMENTS*
        sizeof(float), NULL, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[1], 
        mSubParticleVertexDataVbo, cudaGraphicsMapFlagsNone) );

    // create opengl vbo for storing the ids of the particles in 
    // default state
    glGenBuffers(1, &mParticleIdsDefaultVbo);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mParticleIdsDefaultVbo);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, mParameters.numParticles*sizeof(int),
        NULL, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[2], 
        mParticleIdsDefaultVbo, cudaGraphicsMapFlagsNone) );

    // create opengl vbo for storing the ids of the active sub particles 
    glGenBuffers(1, &mSubParticleIdsVbo);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mSubParticleIdsVbo);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, mParameters.numParticles*sizeof(int)*8,
        NULL, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[3], 
        mSubParticleIdsVbo, cudaGraphicsMapFlagsNone) );

    CUDA_SAFE_CALL( hipMalloc(&mSubParticleSimulationDataDevPtr, 
        8*mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS) );
    CUDA_SAFE_CALL( hipMemset(mSubParticleSimulationDataDevPtr, 0,
        8*mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleHashsDevPtr, 
        8*mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleStatesDevPtr, 
               mParameters.numParticles*sizeof(char)) );
    CUDA_SAFE_CALL( hipMemset(mParticleStatesDevPtr, 0, 
        mParameters.numParticles*sizeof(char)) );
    mParticleStates = new char[mParameters.numParticles];    
    CUDA_SAFE_CALL( hipMalloc(&_isSplitDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&_isBoundaryDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&_isDefaultDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&_splitPrefixSumDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );    
    CUDA_SAFE_CALL( hipMalloc(&_boundaryPrefixSumDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );    
    CUDA_SAFE_CALL( hipMalloc(&_defaultPrefixSumDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );    
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsDefaultDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsBoundaryDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsSplitDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleCellStartIdsDevPtr, 
        sizeof(int)*mParameters.gridDimSubParticles[0]*
        mParameters.gridDimSubParticles[1]*
        mParameters.gridDimSubParticles[2]) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleCellEndIdsDevPtr, 
        sizeof(int)*mParameters.gridDimSubParticles[0]*
        mParameters.gridDimSubParticles[1]*
        mParameters.gridDimSubParticles[2]) );
}
//-----------------------------------------------------------------------------
void ParticleSimulation::Bind () const 
{
    // copy simulation parameters to constant memory on device.
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gSimParamsDev), (void*)&mParameters, 
        sizeof(SimulationParameters)) );  
}
//-----------------------------------------------------------------------------

void ParticleSimulation::Advance ()
{
    this->map();
    this->computeParticleHash();
    this->computeSubParticleHash();
    this->sortParticleIdsByHash();
    this->sortSubParticleIdsByHash();
    this->computeCellStartEndList();
    this->computeSubParticleCellStartEndList ();
    this->computeDensityPressure();
    this->computeAcceleration();
    this->integrate();
    this->integrateSubParticles();
    this->handleCollisions();
    this->computeParticleState();
    this->collect();
    this->initializeSubParticles();
    this->unmap();
}
//-----------------------------------------------------------------------------
float ParticleSimulation::GetParticleRadius () const
{
    return powf((3.0*mParameters.fluidVolume)/
        (4.0*M_PI*mParameters.numParticles), 1.0f/3.0f);
}
//-----------------------------------------------------------------------------
float ParticleSimulation::GetSubParticleRadius () const
{
    return 0.5f*this->GetParticleRadius();
}
//-----------------------------------------------------------------------------
const char* ParticleSimulation::GetParticleState () const
{
    CUDA_SAFE_CALL( hipMemcpy(mParticleStates, mParticleStatesDevPtr, 
        sizeof(char)*mParameters.numParticles, hipMemcpyDeviceToHost) );
    return mParticleStates;
}
//-----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumParticles () const
{
    return mParameters.numParticles;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::SetNPartThresh (float dVal)
{
    mParameters.nPartTresh += dVal;
    printf("# particle thresh %f\n", mParameters.nPartTresh);
    this->Bind();
}
//-----------------------------------------------------------------------------
void ParticleSimulation::DecreaseCmDistanceThresh ()
{
    _rightI = mParameters.cmDistanceThresh;
    mParameters.cmDistanceThresh = 0.5f*(_rightI - _leftI);
    printf("cmDistance = %f\n", mParameters.cmDistanceThresh);
    this->Bind();
}
//-----------------------------------------------------------------------------
void ParticleSimulation::IncreaseCmDistanceThresh ()
{
    _leftI = mParameters.cmDistanceThresh;
    mParameters.cmDistanceThresh = 0.5f*(_rightI - _leftI);
    printf("cmDistance = %f\n", mParameters.cmDistanceThresh);
    this->Bind();
}
//-----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLParticleVertexBufferObject () const
{
    return mParticleVertexDataVbo;
}
//-----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLParticleIndexVertexBufferObject () const
{
    return mParticleIdsDefaultVbo;
}
//-----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumParticlesDefault () const
{
    return mNumParticlesDefault;
}
//----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLSubParticleVertexBufferObject () const
{
    return mSubParticleVertexDataVbo;
}
//----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLSubParticleIndexVertexBufferObject () const
{
    return mSubParticleIdsVbo;
}
//----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumSubParticles () const
{
    return mNumSubParticles;
}
//----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumSubParticlesRegular () const
{
    return mNumParticlesSplit*8;
}
//----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumSubParticlesBoundary () const
{
    return mNumParticlesBoundary*8;
}
//-----------------------------------------------------------------------------
// Definition of private methods
//-----------------------------------------------------------------------------
void ParticleSimulation::computeParticleHash () 
{
    compute_particle_hash <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleIdsDevPtr, 
        mParticleHashListDevPtr, mParameters.numParticles);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeSubParticleHash ()
{
    if (mNumSubParticles != 0)
    {
        compute_sub_particle_hash
            <<< mNumBlocksSubParticle, mThreadsPerBlockSubParticle >>> 
            (mSubParticleVertexDataDevPtr, mSubParticleIdsDevPtr, 
            mSubParticleHashsDevPtr, mNumSubParticles);
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::sortParticleIdsByHash ()
{
   thrust::sort_by_key(thrust::device_ptr<int>(mParticleHashListDevPtr),
        thrust::device_ptr<int>(mParticleHashListDevPtr + 
        mParameters.numParticles), 
        thrust::device_ptr<int>(mParticleIdsDevPtr));
}
//-----------------------------------------------------------------------------
void ParticleSimulation::sortSubParticleIdsByHash ()
{
    thrust::sort_by_key(thrust::device_ptr<int>(mSubParticleHashsDevPtr),
        thrust::device_ptr<int>(mSubParticleHashsDevPtr + 
        mNumSubParticles),
        thrust::device_ptr<int>(mSubParticleIdsDevPtr));
}

//-----------------------------------------------------------------------------
void ParticleSimulation::computeCellStartEndList () 
{
    int* dim = mParameters.gridDim; 
    unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(int);

    hipMemset(mCellStartListDevPtr, EMPTY_CELL, size);
    hipMemset(mCellEndListDevPtr, EMPTY_CELL, size);
    
    int sharedMemSize = sizeof(int)*(mThreadsPerBlock + 1);
    compute_cell_start_end <<< mNumBlocks, mThreadsPerBlock,  sharedMemSize>>>  
        (mParticleHashListDevPtr, mCellStartListDevPtr, 
        mCellEndListDevPtr, mParameters.numParticles);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeSubParticleCellStartEndList () 
{
    int* dim = mParameters.gridDimSubParticles; 
    unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(int);

    if (mNumSubParticles == 0)
    {
        return;
    }

    hipMemset(mSubParticleCellStartIdsDevPtr, EMPTY_CELL, size);
    hipMemset(mSubParticleCellEndIdsDevPtr, EMPTY_CELL, size);
    
    int sharedMemSize = sizeof(int)*(mThreadsPerBlockSubParticle + 1);
    compute_cell_start_end 
        <<< mNumBlocksSubParticle, mThreadsPerBlockSubParticle, sharedMemSize>>>  
        (mSubParticleHashsDevPtr, mSubParticleCellStartIdsDevPtr, 
        mSubParticleCellEndIdsDevPtr, mNumSubParticles);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeDensityPressure () 
{
    compute_particle_density_pressure <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr,mParticleSimulationDataDevPtr, 
        mParticleIdsDevPtr, mCellStartListDevPtr, mCellEndListDevPtr);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeAcceleration ()
{
    compute_particle_acceleration_ifsurf <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr, 
        mParticleIdsDevPtr, mCellStartListDevPtr, mCellEndListDevPtr,
        mIsSurfaceParticleDevPtr);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeAccelerationSubParticles ()
{
}
//-----------------------------------------------------------------------------
void ParticleSimulation::integrate ()
{
    integrate_euler <<< mNumBlocks, mThreadsPerBlock >>>
        (mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::integrateSubParticles ()
{
    if (mNumSubParticles != 0)
    {
        integrate_sub_particles_euler 
            <<< mNumBlocksSubParticle, mThreadsPerBlockSubParticle >>>
            (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr,
            mSubParticleIdsDevPtr, mNumSubParticles);
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::handleCollisions ()
{
    collision_handling <<< mNumBlocks, mThreadsPerBlock >>>
        (mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeParticleState ()
{
    shift_state <<< mNumBlocks, mThreadsPerBlock >>> (mParticleStatesDevPtr);
    update_particle_state <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleStatesDevPtr, mParticleIdsDevPtr,
        mCellStartListDevPtr, mCellEndListDevPtr);

}
//-----------------------------------------------------------------------------
// Collects all id's of active sub particles, particles with state "split",
// particles with state "boundary", particles with that "default" in their own
// arrays and computes the total number of each particle.
void ParticleSimulation::collect ()
{    
    CUDA_SAFE_CALL( hipMemset(_isSplitDevPtr, 0, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMemset(_isBoundaryDevPtr, 0, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMemset(_isDefaultDevPtr, 0, 
        (mParameters.numParticles + 1)*sizeof(int)) );

    check_split_boundary_default <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleStatesDevPtr, _isSplitDevPtr, _isBoundaryDevPtr, 
        _isDefaultDevPtr);

    thrust::exclusive_scan(thrust::device_ptr<int>(_isSplitDevPtr),
        thrust::device_ptr<int>(_isSplitDevPtr + mParameters.numParticles
        + 1), thrust::device_ptr<int>(_splitPrefixSumDevPtr));
    thrust::exclusive_scan(thrust::device_ptr<int>(_isBoundaryDevPtr),
        thrust::device_ptr<int>(_isBoundaryDevPtr + mParameters.numParticles
        + 1), thrust::device_ptr<int>(_boundaryPrefixSumDevPtr));
    thrust::exclusive_scan(thrust::device_ptr<int>(_isDefaultDevPtr),
        thrust::device_ptr<int>(_isDefaultDevPtr + mParameters.numParticles
        + 1), thrust::device_ptr<int>(_defaultPrefixSumDevPtr));

    CUDA_SAFE_CALL( hipMemcpy(&mNumParticlesSplit,
        &_splitPrefixSumDevPtr[mParameters.numParticles], sizeof(int), 
        hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(&mNumParticlesBoundary,
        &_boundaryPrefixSumDevPtr[mParameters.numParticles], sizeof(int), 
        hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(&mNumParticlesDefault,
        &_defaultPrefixSumDevPtr[mParameters.numParticles], sizeof(int), 
        hipMemcpyDeviceToHost) );    

    collect_ids <<< mNumBlocks, mThreadsPerBlock >>>
        (mSubParticleIdsDevPtr, mParticleIdsSplitDevPtr,
        mParticleIdsBoundaryDevPtr, mParticleIdsDefaultDevPtr, 
        _isSplitDevPtr, _isBoundaryDevPtr, _isDefaultDevPtr, 
        _splitPrefixSumDevPtr, _boundaryPrefixSumDevPtr, 
        _defaultPrefixSumDevPtr, mNumParticlesSplit);

    mNumSubParticles = 8*(mNumParticlesSplit + mNumParticlesBoundary);

    // compute how many cuda blocks and how many threads a block are needed for
    // split particles, boundary particles, default particles.
    compute_particle_kernel_invocation_information(mThreadsPerBlockSplit, 
        mNumBlocksSplit, mNumParticlesSplit);
    compute_particle_kernel_invocation_information(mThreadsPerBlockBoundary, 
        mNumBlocksBoundary, mNumParticlesBoundary);
    compute_particle_kernel_invocation_information(mThreadsPerBlockDefault, 
        mNumBlocksDefault, mNumParticlesDefault);
    compute_particle_kernel_invocation_information(mThreadsPerBlockSubParticle,
        mNumBlocksSubParticle, mNumSubParticles);
}
//-----------------------------------------------------------------------------
//  initializes new sub particles if a parent particle has changed its state 
//  from "default" to "boundary" or "split"
void ParticleSimulation::initializeSubParticles () 
{
    if (mNumParticlesSplit > 0)
    {
        initialize_sub_particles <<<mNumBlocksSplit, mThreadsPerBlockSplit>>> 
            (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr,
            mParticleIdsSplitDevPtr, mParticleVertexDataDevPtr, 
            mParticleSimulationDataDevPtr, mParticleStatesDevPtr, 
            mNumParticlesSplit);    
    }

    if (mNumParticlesBoundary > 0)
    {
        initialize_sub_particles <<<mNumBlocksBoundary, mThreadsPerBlockBoundary>>> 
            (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr,
            mParticleIdsBoundaryDevPtr, mParticleVertexDataDevPtr, 
            mParticleSimulationDataDevPtr, mParticleStatesDevPtr, 
            mNumParticlesBoundary);
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::map () 
{
    hipGraphicsMapResources(4, mGraphicsResources);
    size_t nBytes;
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mParticleVertexDataDevPtr), &nBytes,
        mGraphicsResources[0]);
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mSubParticleVertexDataDevPtr), &nBytes,
        mGraphicsResources[1]);
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mParticleIdsDefaultDevPtr), &nBytes,
        mGraphicsResources[2]);
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mSubParticleIdsDevPtr), &nBytes,
        mGraphicsResources[3]);
}

void ParticleSimulation::unmap () 
{
    hipGraphicsUnmapResources(4, mGraphicsResources);
    //cudaGLUnmapBufferObject(mParticleVertexDataVbo);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::SaveInfoTable (const std::string& filename) 
{
    using namespace std;

    ofstream file;

    file.open(filename);

    int* pIdList = new int[mParameters.numParticles];
    int* pHashList = new int[mParameters.numParticles];

    int cellListSize = mParameters.gridDim[0]*mParameters.gridDim[1]*
        mParameters.gridDim[2];

    int* pCellStartList = new int[cellListSize];
    int* pCellEndList = new int[cellListSize];

    //this->map();

    hipMemcpy(pHashList, mParticleHashListDevPtr, 
        mParameters.numParticles*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pIdList, mParticleIdsDevPtr, 
        mParameters.numParticles*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pCellStartList, mCellStartListDevPtr, 
        cellListSize*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pCellEndList, mCellEndListDevPtr, 
        cellListSize*sizeof(int), hipMemcpyDeviceToHost);
    
    file << "Number of particles " << mParameters.numParticles << endl; 
    file << setw(8) << "index" << setw(12) << " id" << setw(12) << 
        " hash" << setw(12) << " start" << setw(12) << " end" << endl;

    for (unsigned int i = 0; i < cellListSize; i++) 
    {
        file << setw(8) << i;

        if(i < mParameters.numParticles)
        {
            file << setw(12) << pIdList[i];
            file << setw(12) << pHashList[i];
        } 
        else 
        {
            file << setw(12) << "";
            file << setw(12) << "";
        }

        if(pCellStartList[i] == EMPTY_CELL) 
        {
            file << setw(12) << "";
        } 
        else
        {
            file << setw(12) << pCellStartList[i];
        }
        
        if(pCellEndList[i] == EMPTY_CELL)
        {
            file << setw(12) << "" << endl;
        } 
        else
        {
            file << setw(12) << pCellEndList[i] << endl;
        }
    }

    delete[] pIdList;
    delete[] pHashList;
    delete[] pCellStartList;
    delete[] pCellEndList;

    file.close();

    //this->unmap();
}
//-----------------------------------------------------------------------------
void ParticleSimulation::SaveParticleInfo (const std::string& filename)
{
    using namespace std;

    this->map();

    ofstream file;
    
    file.open(filename);

    float* particleVertexData = 
        new float[VD_NUM_ELEMENTS*mParameters.numParticles]; 
    float* particleSimulationData = 
        new float[SD_NUM_ELEMENTS*mParameters.numParticles]; 

    // copy particle information from device to host
    hipMemcpy(particleVertexData, mParticleVertexDataDevPtr, 
		VD_NUM_ELEMENTS*mParameters.numParticles*sizeof(float), 
		hipMemcpyDeviceToHost);
    hipMemcpy(particleSimulationData, mParticleSimulationDataDevPtr, 
		SD_NUM_ELEMENTS*mParameters.numParticles*sizeof(float), 
		hipMemcpyDeviceToHost);    
    
    // set max. chars for each column of the table
    int columnWidth = 20;
    
    file << setw(columnWidth) << "Index";
    file << setw(columnWidth) << "X";
    file << setw(columnWidth) << "Y";
    file << setw(columnWidth) << "Z";
    file << setw(columnWidth) << "Density";
    file << setw(columnWidth) << "Pressure";
    file << setw(columnWidth) << "Acc X";
    file << setw(columnWidth) << "Acc Y";
    file << setw(columnWidth) << "Acc Z";
    file << endl;

    for (unsigned int i = 0; i < mParameters.numParticles; i++) 
    {
        file << setw(columnWidth) << i;
        file << setw(columnWidth) 
            << particleVertexData[VD_NUM_ELEMENTS*i + VD_POS_X];
        file << setw(columnWidth) 
            << particleVertexData[VD_NUM_ELEMENTS*i + VD_POS_Y];
        file << setw(columnWidth) 
            << particleVertexData[VD_NUM_ELEMENTS*i + VD_POS_Z];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_DENSITY];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_PRESSURE];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_ACC_X];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_ACC_Y];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_ACC_Z];
        // TODO: rest of the params.
        file << endl;
    }


    delete[] particleVertexData;
    delete[] particleSimulationData;

    file.close();

    this->unmap();
}
//-----------------------------------------------------------------------------
/*
unsigned int ParticleSimulation::GetSizeMemoryGPU () const
{
    float size;
    
    size += mNumPar


}*/
//-----------------------------------------------------------------------------
//  definition of aux. functions
//-----------------------------------------------------------------------------
// Creates a set of particles, that are aligned in a cube, given the starting
// point of the box [sx, sy, sz] the length of the cube in each direction [d]
// and the approximate amount of total particles [numParticles].
//
// Returns a pointer to the vertex data of the particles in [particleVD] and
// a pointer to the simulation data of the particles in [particleSD] and the
// actual amount of particles created.
void create_particle_box (float sx, float sy, float sz, float d, 
    unsigned int numParticles, float** particleVD, float** particleSD,
    unsigned int* numParticlesCreated)
{
    // computed number of particles in each direction
    unsigned int num = pow(static_cast<double>(numParticles), 1.0/3.0);
    *numParticlesCreated = num*num*num;

    *particleVD = new float[*numParticlesCreated*VD_NUM_ELEMENTS];
    *particleSD = new float[*numParticlesCreated*SD_NUM_ELEMENTS];

    // check if new failed.
    if ((*particleSD) == NULL || (*particleSD) == NULL)
    {
        *numParticlesCreated = 0;
        return;
    }

    // compute spatial increment
    float dx = d/static_cast<float>(num - 1);

    // seed the particles inside the cube
    
    // set the position of each particle
    unsigned int idx;
    
    for (unsigned int k = 0; k < num; k++) 
    {
		for (unsigned int j = 0; j < num; j++) 
        {
			for (unsigned int i = 0; i < num; i++) 
            {
			    idx = VD_NUM_ELEMENTS*(num*(num*k+j)+i);
                (*particleVD)[idx + VD_POS_X] = sx + i*dx;
                (*particleVD)[idx + VD_POS_Y] = sy + j*dx;
                (*particleVD)[idx + VD_POS_Z] = sz + k*dx;
            }
		}
	}
    
    // set other particles attributes to 0.0f
    memset((*particleSD), 0, 
        sizeof(float)*SD_NUM_ELEMENTS*(*numParticlesCreated));
}
//-----------------------------------------------------------------------------
// Sets the simulation domain in the [parameters], based on a starting point
// [xs, ys, zs] an ending point [xe, ye, ze] and the distance between two
// grid points [gridSpacing].
void set_simulation_domain (float xs, float ys, float zs, float xe,
    float ye, float ze, float gridSpacing, float gridSpacingSubParticles,
    SimulationParameters* parameters)
{
    parameters->gridOrigin[0] = xs;
    parameters->gridOrigin[1] = ys;
    parameters->gridOrigin[2] = zs;
    parameters->gridDim[0] = static_cast<int>((xe - xs)/gridSpacing + 0.5);
    parameters->gridDim[1] = static_cast<int>((ye - ys)/gridSpacing + 0.5);
    parameters->gridDim[2] = static_cast<int>((ze - zs)/gridSpacing + 0.5);
    parameters->gridDimSubParticles[0] = 
        static_cast<int>((xe - xs)/gridSpacingSubParticles + 0.5);
    parameters->gridDimSubParticles[1] = 
        static_cast<int>((ye - ys)/gridSpacingSubParticles + 0.5);
    parameters->gridDimSubParticles[2] = 
        static_cast<int>((ze - zs)/gridSpacingSubParticles + 0.5);
    parameters->gridSpacing = gridSpacing;
    parameters->gridSpacingSubParticles = gridSpacingSubParticles;
}
//-----------------------------------------------------------------------------
void compute_particle_kernel_invocation_information
    (unsigned int& nThreadsBlock, unsigned int& nBlocks, 
    unsigned int numParticles)
{
    if (numParticles == 0)
    {
        nThreadsBlock = 0;
        nBlocks = 0;
        return;
    }

    nThreadsBlock = numParticles > 256 ? 256 : numParticles;
    nBlocks = numParticles % nThreadsBlock == 0 ? numParticles/nThreadsBlock : 
        numParticles/nThreadsBlock + 1;
}