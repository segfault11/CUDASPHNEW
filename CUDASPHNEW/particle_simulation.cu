#include "hip/hip_runtime.h"
#include <thrust\sort.h>
#include <thrust\device_ptr.h>
#include <thrust\for_each.h>
#include <thrust\iterator\zip_iterator.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include "particle_simulation.h"
#include "util.h"
#include "cgtk\include\clock.h"
#include "boundary_map.h"
#include <thrust\scan.h>
#include <stdexcept>
#include "portable_pixmap.h"
#include "arr.h"

using namespace std;

//-----------------------------------------------------------------------------
//  DEVICE CODE 
//-----------------------------------------------------------------------------

//-----------------------------------------------------------------------------
//  global device variables 
//-----------------------------------------------------------------------------
__constant__ SimulationParameters gSimParamsDev;

texture<float, hipTextureType1D, hipReadModeElementType> 
    gParticleVertexData;
texture<float, hipTextureType1D, hipReadModeElementType> 
    gParticleSimulationData;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gCellStartList;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gCellEndList;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gSortedParticleIdList;
texture<int, hipTextureType1D, hipReadModeElementType> 
    gParticleHashList;

//  information about boundary handling
__constant__ float gBoundaryGridOrigin[3];
__constant__ float gBoundaryGridSpacing;
__constant__ unsigned int gBoundaryGridDimensions[3];
__constant__ float gBoundaryGridLength[3];
__constant__ float gBoundaryRestDistance;

texture<float, hipTextureType3D, hipReadModeElementType> 
    gBoundaryDistances;
texture<float, hipTextureType3D, hipReadModeElementType> 
    gBoundaryDensities;
texture<float, hipTextureType3D, hipReadModeElementType> 
    gBoundaryViscosities;

//-----------------------------------------------------------------------------
//  declaration of aux. functions (device) 
//-----------------------------------------------------------------------------
__device__ inline int3 compute_grid_coordinate (float3 pos, float d);
__device__ inline int3 compute_grid_coordinate_sub_particles(float3 pos, 
    float d);
__device__ inline int compute_hash_from_grid_coordinate (int i, int j, int k);
__device__ inline int compute_hash_from_grid_coordinate_sub_particle (int i, 
    int j, int k);
__device__ inline float compute_distance (float3 a, float3 b);
__device__ inline float compute_squared_distance (float3 a, float3 b);
__device__ inline float norm (const float3& a);
__device__ inline void normalize (float3& a);
__device__ inline float dot_product (const float3& a, const float3& b);
__device__ float compute_particle_density_cell (const float3 &pos, 
	float* pParticleList, int* pParticleIdList, int start, int end);
__device__ float compute_sub_particle_density_cell (const float3 &pos, 
	float* particleVertexData, int* particleIdList, int start, int end);
__device__ inline void compute_viscosity_pressure_forces_and_ifsurf_cell
    (const float3& xi, float rhoi, float pi, const float3& vi,
    float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int start, int end, 
    float3* pressureForce, float3* viscosityForce, float3* colGra, 
    float* colLapl, float3* sumPosNeighbor, float* nNeighbors);
__device__ inline void compute_sub_particle_viscosity_pressure_forces_cell
    (const float3& xi, float rhoi, float pi, const float3& vi,
    float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int start, int end, 
    float3* force, float3* colGra, float* colLapl);
__device__ inline void project_quantities_cell (float3& acc, float& density,
    float& pressure, float& numNeighbors, const float3& xi, int start, int end);
//-----------------------------------------------------------------------------
// CUDA Kernel definitions 
//-----------------------------------------------------------------------------
__global__ void compute_particle_hash (float* particleVertexData, 
    int* particleIdList, int* particleHashList, unsigned int numParticles) 
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    // calculate corresponding gridpoint
    int x = (int)((tex1Dfetch(gParticleVertexData, idx*VD_NUM_ELEMENTS + VD_POS_X) - 
		gSimParamsDev.gridOrigin[0])/gSimParamsDev.gridSpacing);
    int y = (int)((tex1Dfetch(gParticleVertexData, idx*VD_NUM_ELEMENTS + VD_POS_Y) - 
		gSimParamsDev.gridOrigin[1])/gSimParamsDev.gridSpacing);
    int z = (int)((tex1Dfetch(gParticleVertexData, idx*VD_NUM_ELEMENTS + VD_POS_Z) - 
		gSimParamsDev.gridOrigin[2])/gSimParamsDev.gridSpacing);

    // wrap outer particles to grid
    // TODO: modulo operation using "&" is faster, requires grid dims of 
	// power of two
    x = x % gSimParamsDev.gridDim[0];
    y = y % gSimParamsDev.gridDim[1];
    z = z % gSimParamsDev.gridDim[2];
    
    // calculate hash, i.e. grid cell id
    int hash = gSimParamsDev.gridDim[0]*(gSimParamsDev.gridDim[1]*z + y) + x;

    particleIdList[idx] = idx;
    particleHashList[idx] = hash;
}
//-----------------------------------------------------------------------------
__global__ void compute_sub_particle_hash (float* particleVertexData, 
    int* particleIdList, int* particleHashList, unsigned int numParticles) 
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    int id = particleIdList[idx];

    // calculate corresponding gridpoint
    int x = (int)((particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X] - 
        gSimParamsDev.gridOrigin[0])/gSimParamsDev.gridSpacingSubParticles);
    int y = (int)((particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y] - 
		gSimParamsDev.gridOrigin[1])/gSimParamsDev.gridSpacingSubParticles);
    int z = (int)((particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z] - 
		gSimParamsDev.gridOrigin[2])/gSimParamsDev.gridSpacingSubParticles);

    // wrap outer particles to grid
    // TODO: modulo operation using "&" is faster, requires grid dims of 
	// power of two
    x = x % gSimParamsDev.gridDimSubParticles[0];
    y = y % gSimParamsDev.gridDimSubParticles[1];
    z = z % gSimParamsDev.gridDimSubParticles[2];
    
    // calculate hash, i.e. grid cell id
    int hash = gSimParamsDev.gridDimSubParticles[0]*
        (gSimParamsDev.gridDimSubParticles[1]*z + y) + x;

    particleHashList[idx] = hash;
}
//-----------------------------------------------------------------------------
__global__ void compute_cell_start_end (int* particleHashList, 
	int* cellStartList,  int* cellEndList, unsigned int numParticles)
{
    extern __shared__ int sharedHash[];
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    int hash;

    if (idx < numParticles) 
    {
        hash = particleHashList[idx];
        sharedHash[threadIdx.x + 1] = hash;
        
        if (idx > 0 && threadIdx.x == 0) 
        {
            sharedHash[0] = particleHashList[idx - 1];
        }
    }

    __syncthreads();

    if (idx < numParticles) 
    {
        if (idx == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStartList[hash] = idx;
        
            if (idx > 0) 
            {
                cellEndList[sharedHash[threadIdx.x]] = idx;
            }
        }

        if (idx == numParticles - 1)
        {
            cellEndList[hash] = idx + 1;
        }
    }
}
//-----------------------------------------------------------------------------
//  Compute density and pressure for each particle 
__global__ void compute_particle_density_pressure (float* particleVertexData, 
	float* particleSimulationData, int* particleIdList, int* cellStartList, 
    int* cellEndList) 
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= gSimParamsDev.numParticles) {
        return;
    }

	int id = particleIdList[idx];

	float density = 0.0f;
    float pressure;
    float3 pos;

    // get particles position form vertex data
    pos.x = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_X);
    pos.y = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Y);
    pos.z = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Z);

    int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupport);
    int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupport);

    int hash;
    int start;
    int end;
    
    // compute density contribution from neighbor particles
    for(int k = c0.z; k <= c1.z; k++) 
    {
        for(int j = c0.y; j <= c1.y; j++) 
        {
            for(int i = c0.x; i <= c1.x; i++)
            {
                hash = compute_hash_from_grid_coordinate(i, j, k);
                start = tex1Dfetch(gCellStartList, hash);
                end = tex1Dfetch(gCellEndList, hash);
                density += compute_particle_density_cell(pos, 
                    particleVertexData, particleIdList, start, end);
            }
        }
    }
    
    density *= gSimParamsDev.particleMass;

    // compute density contribution from the wall
    float u = (pos.x - gBoundaryGridOrigin[0])/gBoundaryGridLength[0];
    float v = (pos.y - gBoundaryGridOrigin[1])/gBoundaryGridLength[1];
    float w = (pos.z - gBoundaryGridOrigin[2])/gBoundaryGridLength[2];
    float densWall = tex3D(gBoundaryDensities, u, v, w);

    density += densWall;

    pressure = gSimParamsDev.gasStiffness*(density - 
        gSimParamsDev.restDensity);

    // set density and pressure
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_DENSITY] = density;
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_PRESSURE] = pressure; 
}
//-----------------------------------------------------------------------------
//  Compute density and pressure for each sub particle 
__global__ void compute_sub_particle_density_pressure 
    (float* subParticleVertexData, float* subParticleSimulationData, 
    int* particleIdList, int* particleSortedIdList, int* cellStartList, 
    int* cellEndList, unsigned int numParticles) 
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= numParticles)
     {
        return;
    }

	int id = particleIdList[idx];

	float density = 0.0f;
    float pressure;
    float3 pos;

    // get particles position form vertex data
    pos.x = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];
    pos.y = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
    pos.z = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

    int3 c0 = compute_grid_coordinate_sub_particles(pos, 
        -gSimParamsDev.compactSupportSub);
    int3 c1 = compute_grid_coordinate_sub_particles(pos, 
        gSimParamsDev.compactSupportSub);

    int hash;
    int start;
    int end;

    for(int k = c0.z; k <= c1.z; k++) 
    {
        for(int j = c0.y; j <= c1.y; j++) 
        {
            for(int i = c0.x; i <= c1.x; i++)
            {
                hash = compute_hash_from_grid_coordinate_sub_particle(i, j, k);
                start = cellStartList[hash];
                end = cellEndList[hash];
                density += compute_sub_particle_density_cell(pos, 
                    subParticleVertexData, particleSortedIdList, start, end);
            }
        }
    }
    
    density *= gSimParamsDev.subParticleMass;
    pressure = gSimParamsDev.gasStiffness*(density - 
        gSimParamsDev.restDensity);

    // set density and pressure
    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_DENSITY] = density;
    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_PRESSURE] = pressure; 
}
//-----------------------------------------------------------------------------
__global__ void compute_particle_acceleration_ifsurf 
    (float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int* cellStartList,
    int* cellEndList, int* isSurfaceParticle)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }
    
    int id = tex1Dfetch(gSortedParticleIdList, idx);

    float density  = tex1Dfetch(gParticleSimulationData, id*SD_NUM_ELEMENTS
        + SD_DENSITY);
    float pressure = tex1Dfetch(gParticleSimulationData, id*SD_NUM_ELEMENTS
        + SD_PRESSURE);
    float tenCoeff = gSimParamsDev.tensionCoefficient;

    float3 pos;
    pos.x = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_X);
    pos.y = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Y);
    pos.z = tex1Dfetch(gParticleVertexData, id*VD_NUM_ELEMENTS + VD_POS_Z);

    float3 vel;
    vel.x = tex1Dfetch(gParticleSimulationData, 
        id*SD_NUM_ELEMENTS + SD_VEL0_X);
    vel.y = tex1Dfetch(gParticleSimulationData, 
        id*SD_NUM_ELEMENTS + SD_VEL0_Y);
    vel.z = tex1Dfetch(gParticleSimulationData, 
        id*SD_NUM_ELEMENTS + SD_VEL0_Z);

    int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupport);
    int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupport);

    float3 force;
    force.x = 0.0f;
    force.y = 0.0f;
    force.z = 0.0f;

    float3 pressureForce;
    pressureForce.x = 0.0f;
    pressureForce.y = 0.0f;
    pressureForce.z = 0.0f;

    float3 viscosityForce;
    viscosityForce.x = 0.0f;
    viscosityForce.y = 0.0f;
    viscosityForce.z = 0.0f;

    float3 colGra;
    colGra.x = 0.0f;
    colGra.y = 0.0f;
    colGra.z = 0.0f;

    // [sumPosNeigbor] and [nNeigbors] are used to computed the center of mass
    // of the neighborhood of this particle (this also includes the particle
    // itself
    float3 sumPosNeighbor;
    sumPosNeighbor.x = pos.x;
    sumPosNeighbor.x = pos.y;
    sumPosNeighbor.x = pos.z;

    float nNeighbors = 1.0f;

    float colLapl;
    float colGraNorm;
    float grav = gSimParamsDev.gravity;

    int hash;
    int start;
    int end;

    // compute viscosity and pressure forces
    for(int k = c0.z; k <= c1.z; k++)
    {
        for(int j = c0.y; j <= c1.y; j++)
        {
            for(int i = c0.x; i <= c1.x; i++)
            {
                hash  = compute_hash_from_grid_coordinate(i, j, k);
                start = tex1Dfetch(gCellStartList, hash);
                end = tex1Dfetch(gCellEndList, hash);
                compute_viscosity_pressure_forces_and_ifsurf_cell(pos, density, 
                    pressure, vel, particleVertexData, particleSimulationData,
                    particleIdList, start, end, &pressureForce, &viscosityForce, 
                    &colGra, &colLapl, &sumPosNeighbor, &nNeighbors);
            }
        }
    }
    
    // compute distance to wall
    float u = (pos.x - gBoundaryGridOrigin[0])/gBoundaryGridLength[0];
    float v = (pos.y - gBoundaryGridOrigin[1])/gBoundaryGridLength[1];
    float w = (pos.z - gBoundaryGridOrigin[2])/gBoundaryGridLength[2];
    float distWall = tex3D(gBoundaryDistances, u, v, w);
    
    // add viscosity force
    force.x += viscosityForce.x;
    force.y += viscosityForce.y;
    force.z += viscosityForce.z;

    // add pressure force
    force.x += pressureForce.x;
    force.y += pressureForce.y;
    force.z += pressureForce.z;    
    
    float coeff = 1.0f;gSimParamsDev.particleMass/
            (gSimParamsDev.timeStep*gSimParamsDev.timeStep)*
            (distWall - gBoundaryRestDistance);

    if (distWall < gBoundaryRestDistance)
    {
        float dX = gBoundaryGridSpacing/gBoundaryGridLength[0];
        float dY = gBoundaryGridSpacing/gBoundaryGridLength[1];
        float dZ = gBoundaryGridSpacing/gBoundaryGridLength[2];
        float3 graN;

        graN.x = (tex3D(gBoundaryDistances, u + dX, v, w) - 
            tex3D(gBoundaryDistances, u - dX, v, w))/(2*dX);
        graN.y = (tex3D(gBoundaryDistances, u, v + dY, w) - 
            tex3D(gBoundaryDistances, u, v - dY, w))/(2*dY);
        graN.z = (tex3D(gBoundaryDistances, u, v, w + dZ) - 
            tex3D(gBoundaryDistances, u, v, w - dZ))/(2*dZ);
        normalize(graN);

        // in boundary handling case just, add the pressure force to the force
        force.x += coeff*graN.x;
        force.y += coeff*graN.y;
        force.z += coeff*graN.z;   
        
        // viscosity contribution of the wall
        float visWallCoeff = tex3D(gBoundaryViscosities, u, v, w);
        force.x -= vel.x*visWallCoeff;
        force.y -= vel.y*visWallCoeff;
        force.z -= vel.z*visWallCoeff;
    } 
    else
    {
        // add surface tension force
        colGraNorm = sqrtf(colGra.x*colGra.x + colGra.y*colGra.y 
            + colGra.z*colGra.z);

        float fCoeff = tenCoeff*colLapl/colGraNorm;

        if(colGraNorm > gSimParamsDev.normThresh) 
        {
            force.x -= fCoeff*colGra.x;
            force.y -= fCoeff*colGra.y;
            force.z -= fCoeff*colGra.z;
        }    
    }

    // store the actual acceleration
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_X] = force.x/density;  
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_Y] = force.y/density
        - grav;  
    particleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_Z] = force.z/density;  
}

//-----------------------------------------------------------------------------
__global__ void compute_sub_particle_acceleration
    (float* subParticleVertexData, float* subParticleSimulationData, 
    int* subParticleIdList, int* subParticleSortedIdList, int* cellStartList,
    int* cellEndList, unsigned int numParticles)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }
    
    int id = subParticleIdList[idx];

    float density  = subParticleSimulationData[id*SD_NUM_ELEMENTS 
        + SD_DENSITY];
    float pressure = subParticleSimulationData[id*SD_NUM_ELEMENTS 
        + SD_PRESSURE];
    float tenCoeff = gSimParamsDev.tensionCoefficient;

    float3 pos;
    pos.x = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];
    pos.y = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
    pos.z = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

    float3 vel;
    vel.x = subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_X];
    vel.y = subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_Y];
    vel.z = subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_Z];

    int3 c0 = compute_grid_coordinate_sub_particles(pos, 
        -gSimParamsDev.compactSupportSub);
    int3 c1 = compute_grid_coordinate_sub_particles(pos, 
        gSimParamsDev.compactSupportSub);

    float3 force;
    force.x = 0.0f;
    force.y = 0.0f;
    force.z = 0.0f;

    float3 colGra;
    colGra.x = 0.0f;
    colGra.y = 0.0f;
    colGra.z = 0.0f;

    float colLapl;
    float colGraNorm;
    float grav = gSimParamsDev.gravity;

    int hash;
    int start;
    int end;

    // compute viscosity and pressure forces
    for (int k = c0.z; k <= c1.z; k++)
    {
        for (int j = c0.y; j <= c1.y; j++)
        {
            for (int i = c0.x; i <= c1.x; i++)
            {
                hash  = compute_hash_from_grid_coordinate_sub_particle(i,
                    j, k);

                start = cellStartList[hash];
                end = cellEndList[hash];
                compute_sub_particle_viscosity_pressure_forces_cell(pos, 
                    density, pressure, vel, subParticleVertexData, 
                    subParticleSimulationData, subParticleSortedIdList, 
                    start, end, &force, &colGra, &colLapl);
            }
        }
    }

    // surface tension
    colGraNorm = sqrtf(colGra.x*colGra.x + colGra.y*colGra.y 
        + colGra.z*colGra.z);

    float fCoeff = tenCoeff*colLapl/colGraNorm;

    if (colGraNorm > gSimParamsDev.normThresh) 
    {
        force.x -= fCoeff*colGra.x;
        force.y -= fCoeff*colGra.y;
        force.z -= fCoeff*colGra.z;
    }
    
    // store the actual acceleration
    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_X] = force.x/density;  
    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_Y] = force.y/density
        - grav;  
    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_ACC_Z] = force.z/density;  
}
//----------------------------------------------------------------------------
__global__ void project_quantities (float* subParticleVertexData, 
    float* subParticleSimulationData, float* particleVertexData, 
    float* particleSimulationData, int* subParticleIds, 
    unsigned int numParticles, unsigned int offset)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    int id = subParticleIds[idx + offset];
        
    float3 pos;
    pos.x = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];
    pos.y = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
    pos.z = subParticleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

    int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupportSub);
    int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupportSub);

    float3 acc;
    acc.x = 0.0f;
    acc.y = 0.0f;
    acc.z = 0.0f;
    float density = 0.0f;
    float pressure = 0.0f;
    float numNeighbours = 0.0f;

    int hash;
    int start;
    int end;

    // compute viscosity and pressure forces
    for(int k = c0.z; k <= c1.z; k++)
    {
        for(int j = c0.y; j <= c1.y; j++)
        {
            for(int i = c0.x; i <= c1.x; i++)
            {
                hash  = compute_hash_from_grid_coordinate(i, j, k);
                start = tex1Dfetch(gCellStartList, hash);
                end = tex1Dfetch(gCellEndList, hash);
                project_quantities_cell(acc, density, pressure, numNeighbours,
                    pos, start, end);
            }
        }
    }

    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_DENSITY] = density/numNeighbours;
    subParticleSimulationData[id*SD_NUM_ELEMENTS + SD_PRESSURE] = pressure/numNeighbours;
}
//----------------------------------------------------------------------------
/*__global__ void compute_sub_particle_acceleration
    (float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int* cellStartList,
    int* cellEndList, int* isSurfaceParticle)
{

}*/
//-----------------------------------------------------------------------------
__global__ void integrate_euler (float* particleVertexData, 
    float* particleSimulationData)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int idVert = idx*VD_NUM_ELEMENTS;
    unsigned int idSim = idx*SD_NUM_ELEMENTS;
    float dt = gSimParamsDev.timeStep;

    particleSimulationData[idSim + SD_VEL0_X] += 
        dt*particleSimulationData[idSim + SD_ACC_X];
    particleSimulationData[idSim + SD_VEL0_Y] += 
        dt*particleSimulationData[idSim + SD_ACC_Y];
    particleSimulationData[idSim + SD_VEL0_Z] += 
        dt*particleSimulationData[idSim + SD_ACC_Z];

    particleVertexData[idVert + VD_POS_X] += 
        dt*particleSimulationData[idSim + SD_VEL0_X];
    particleVertexData[idVert + VD_POS_Y] += 
        dt*particleSimulationData[idSim + SD_VEL0_Y];
    particleVertexData[idVert + VD_POS_Z] += 
        dt*particleSimulationData[idSim + SD_VEL0_Z];   

    // compute density contribution from the wall
   /*float u = (pos.x - gBoundaryGridOrigin[0])/gBoundaryGridLength[0];
    float v = (pos.y - gBoundaryGridOrigin[1])/gBoundaryGridLength[1];
    float w = (pos.z - gBoundaryGridOrigin[2])/gBoundaryGridLength[2];
    float distWall = tex3D(gBoundaryDistances, u, v, w);*/
}
//-----------------------------------------------------------------------------
__global__ void integrate_sub_particles_euler (float* subParticleVertexData, 
    float* subParticleSimulationData, int* subParticleIds,
    unsigned int nSubParticles, unsigned int offset)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= nSubParticles)
    {
        return;
    }

    int id = subParticleIds[idx + offset];
    unsigned int idVert = id*VD_NUM_ELEMENTS;
    unsigned int idSim = id*SD_NUM_ELEMENTS;
    float dt = gSimParamsDev.timeStepSubParticles;
    
    subParticleSimulationData[idSim + SD_VEL0_X] += 
        dt*subParticleSimulationData[idSim + SD_ACC_X];
    subParticleSimulationData[idSim + SD_VEL0_Y] += 
        dt*subParticleSimulationData[idSim + SD_ACC_Y];
    subParticleSimulationData[idSim + SD_VEL0_Z] += 
        dt*subParticleSimulationData[idSim + SD_ACC_Z];
    
    subParticleVertexData[idVert + VD_POS_X] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_X];
    subParticleVertexData[idVert + VD_POS_Y] +=
        dt*subParticleSimulationData[idSim + SD_VEL0_Y];
    subParticleVertexData[idVert + VD_POS_Z] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_Z];
}
//-----------------------------------------------------------------------------
__global__ void integrate_boundary_sub_particles_euler 
    (float* subParticleVertexData, float* subParticleSimulationData, 
    int* subParticleIds, unsigned int nSubParticles, unsigned int offset)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= nSubParticles)
    {
        return;
    }

    int id = subParticleIds[idx + offset];
    unsigned int idVert = id*VD_NUM_ELEMENTS;
    unsigned int idSim = id*SD_NUM_ELEMENTS;
    float dt = gSimParamsDev.timeStep;
    
    /*
    subParticleSimulationData[idSim + SD_VEL0_X] += 
        dt*subParticleSimulationData[idSim + SD_ACC_X];
    subParticleSimulationData[idSim + SD_VEL0_Y] += 
        dt*subParticleSimulationData[idSim + SD_ACC_Y];
    subParticleSimulationData[idSim + SD_VEL0_Z] += 
        dt*subParticleSimulationData[idSim + SD_ACC_Z];
    */
    subParticleVertexData[idVert + VD_POS_X] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_X];
    subParticleVertexData[idVert + VD_POS_Y] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_Y];
    subParticleVertexData[idVert + VD_POS_Z] += 
        dt*subParticleSimulationData[idSim + SD_VEL0_Z];
}
//-----------------------------------------------------------------------------
__global__ void shift_state (char* particleState)
{
        unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

        if (idx >= gSimParamsDev.numParticles)
        {
            return;        
        }

        particleState[idx] = (particleState[idx] << 2);
}
//-----------------------------------------------------------------------------
__global__ void collision_handling (float* particleVertexData, 
    float* particleSimulationData)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int idVert = idx*VD_NUM_ELEMENTS;
    unsigned int idSim = idx*SD_NUM_ELEMENTS;

    float3 pos;
    float3 vel;

    pos.x = tex1Dfetch(gParticleVertexData, idVert + VD_POS_X);
    pos.y = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Y);
    pos.z = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Z);

    vel.x = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_X);
    vel.y = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Y);
    vel.z = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Z);

    float3 local;
    float3 diff;
    float3 nrm;

    float dist;
    float depth;

    // compute "distance" to box, if positive the particle
    // is outside the box.

    // compute local position of the particle to the box
    local.x = pos.x - gSimParamsDev.boxCen[0];
    local.y = pos.y - gSimParamsDev.boxCen[1];
    local.z = pos.z - gSimParamsDev.boxCen[2];

    // project local pos to the upper right quadrand and
    // compute difference to the boxDim vec
    diff.x = abs(local.x) - gSimParamsDev.boxDim[0];
    diff.y = abs(local.y) - gSimParamsDev.boxDim[1];
    diff.z = abs(local.z) - gSimParamsDev.boxDim[2];

    dist = max(diff.x, diff.y);
    dist = max(dist, diff.z);
    
    // if the particle lies outside the box, the collision must be handled
    float3 contact;
    
    if (dist > 0.0f)
    {
        // contact point in "box space"
        contact.x = min(gSimParamsDev.boxDim[0], 
            max(-gSimParamsDev.boxDim[0], local.x));
        contact.y = min(gSimParamsDev.boxDim[1],
            max(-gSimParamsDev.boxDim[1], local.y));
        contact.z = min(gSimParamsDev.boxDim[2],
            max(-gSimParamsDev.boxDim[2], local.z));

        // translate to worldspace
        contact.x += gSimParamsDev.boxCen[0];
        contact.y += gSimParamsDev.boxCen[1];
        contact.z += gSimParamsDev.boxCen[2];

        // compute penetration depth
        depth = compute_distance(contact, pos);

        // compute normal
        nrm.x = pos.x - contact.x;
        nrm.y = pos.y - contact.y;
        nrm.z = pos.z - contact.z;
        normalize(nrm);

        float velNorm = norm(vel);
        float dp     = dot_product(nrm, vel);
        float coeff  = (1 + gSimParamsDev.restitution*depth/
            (gSimParamsDev.timeStep*velNorm))*dp;

        vel.x -= coeff*nrm.x;
        vel.y -= coeff*nrm.y;
        vel.z -= coeff*nrm.z;

        particleVertexData[idVert + VD_POS_X] = contact.x;
        particleVertexData[idVert + VD_POS_Y] = contact.y;
        particleVertexData[idVert + VD_POS_Z] = contact.z;

        particleSimulationData[idSim + SD_VEL0_X] = vel.x;
        particleSimulationData[idSim + SD_VEL0_Y] = vel.y;
        particleSimulationData[idSim + SD_VEL0_Z] = vel.z;
    }
}
//-----------------------------------------------------------------------------
__global__ void collision_handling_sub_particles (float* subParticleVertexData, 
    float* subParticleSimulationData, int* subParticleIds, 
    unsigned int numParticles, unsigned int offset)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    int id = subParticleIds[idx + offset];
    unsigned int idVert = id*VD_NUM_ELEMENTS;
    unsigned int idSim = id*SD_NUM_ELEMENTS;

    float3 pos;
    float3 vel;

    pos.x = subParticleVertexData[idVert + VD_POS_X];
    pos.y = subParticleVertexData[idVert + VD_POS_Y];
    pos.z = subParticleVertexData[idVert + VD_POS_Z];

    vel.x = subParticleSimulationData[idSim + SD_VEL0_X];
    vel.y = subParticleSimulationData[idSim + SD_VEL0_Y];
    vel.z = subParticleSimulationData[idSim + SD_VEL0_Z];

    float3 local;
    float3 diff;
    float3 nrm;

    float dist;
    float depth;

    // compute "distance" to box, if positive the particle
    // is outside the box.

    // compute local position of the particle to the box
    local.x = pos.x - gSimParamsDev.boxCen[0];
    local.y = pos.y - gSimParamsDev.boxCen[1];
    local.z = pos.z - gSimParamsDev.boxCen[2];

    // project local pos to the upper right quadrand and
    // compute difference to the boxDim vec
    diff.x = abs(local.x) - gSimParamsDev.boxDim[0];
    diff.y = abs(local.y) - gSimParamsDev.boxDim[1];
    diff.z = abs(local.z) - gSimParamsDev.boxDim[2];

    dist = max(diff.x, diff.y);
    dist = max(dist, diff.z);
    
    // if the particle lies outside the box, the collision must be handled
    float3 contact;
    
    if (dist > 0.0f)
    {
        // contact point in "box space"
        contact.x = min(gSimParamsDev.boxDim[0], 
            max(-gSimParamsDev.boxDim[0], local.x));
        contact.y = min(gSimParamsDev.boxDim[1],
            max(-gSimParamsDev.boxDim[1], local.y));
        contact.z = min(gSimParamsDev.boxDim[2],
            max(-gSimParamsDev.boxDim[2], local.z));

        // translate to worldspace
        contact.x += gSimParamsDev.boxCen[0];
        contact.y += gSimParamsDev.boxCen[1];
        contact.z += gSimParamsDev.boxCen[2];

        // compute penetration depth
        depth = compute_distance(contact, pos);

        // compute normal
        nrm.x = pos.x - contact.x;
        nrm.y = pos.y - contact.y;
        nrm.z = pos.z - contact.z;
        normalize(nrm);

        float velNorm = norm(vel);
        float dp     = dot_product(nrm, vel);
        float coeff  = (1 + gSimParamsDev.restitution*depth/
            (gSimParamsDev.timeStep*velNorm))*dp;

        vel.x -= coeff*nrm.x;
        vel.y -= coeff*nrm.y;
        vel.z -= coeff*nrm.z;

        subParticleVertexData[idVert + VD_POS_X] = contact.x;
        subParticleVertexData[idVert + VD_POS_Y] = contact.y;
        subParticleVertexData[idVert + VD_POS_Z] = contact.z;

        subParticleSimulationData[idSim + SD_VEL0_X] = vel.x;
        subParticleSimulationData[idSim + SD_VEL0_Y] = vel.y;
        subParticleSimulationData[idSim + SD_VEL0_Z] = vel.z;
    }
}
//-----------------------------------------------------------------------------
__global__ void find_split_particles (float* particleVertexData, 
    char* particleState, int* particleIdList, int* cellStartList, 
    int* cellEndList)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int id = particleIdList[idx];
    float3 pos;

    pos.x = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];

    if (pos.x >= 0.2f && pos.x <= 0.5f)
    {
        particleState[id] |= 1;
    }
}
//-----------------------------------------------------------------------------
__global__ void find_boundary_particles (float* particleVertexData, 
    char* particleState, int* particleIdList, int* cellStartList, 
    int* cellEndList)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    unsigned int id = particleIdList[idx];
    unsigned int state = particleState[id] & 3;
    float3 pos;
    float3 xj;
    float3 r;
    float rn;

    if (state == 1)
    {
        // distribute information to neigbors
        pos.x = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];
        pos.y = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
        pos.z = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

        int3 c0 = compute_grid_coordinate(pos, -gSimParamsDev.compactSupport);
        int3 c1 = compute_grid_coordinate(pos, gSimParamsDev.compactSupport);
        
        int hash;
        int start;
        int end;
        
        for(int k = c0.z; k <= c1.z; k++)
        {
            for(int j = c0.y; j <= c1.y; j++)
            {
                for(int i = c0.x; i <= c1.x; i++)
                {
                    hash  = compute_hash_from_grid_coordinate(i, j, k);
                    start = cellStartList[hash];
                    end = cellEndList[hash];
                    
                    for (int u = start; u < end; u++) 
                    {
                        int v = particleIdList[u]; 
                        xj.x = particleVertexData[v*VD_NUM_ELEMENTS + VD_POS_X];
                        xj.y = particleVertexData[v*VD_NUM_ELEMENTS + VD_POS_Y];                         
                        xj.z = particleVertexData[v*VD_NUM_ELEMENTS + VD_POS_Z];
                        r.x = pos.x - xj.x;
                        r.y = pos.y - xj.y;
                        r.z = pos.z - xj.z;
                        rn = r.x*r.x + r.y*r.y + r.z*r.z;

                        if (rn <= gSimParamsDev.compactSupport*
                            gSimParamsDev.compactSupport)
                        {
                            particleState[v] |= 2;
                        }
                    }
                }
            }
        }
    }
}
//-----------------------------------------------------------------------------
// sets the sub particle vertex & simulation data for each particle, that has
// changed its state from "default" to "boundary" or "split".
// This kernel is called for particles that were split and for boundary 
// particles. [numParticles] refers to the total number of those particles.
__global__ void initialize_sub_particles (float* subParticleVertexData, 
    float* subParticleSimulationData, int* particleIds, 
    float* particleVertexData, float* particleSimulationData, 
    char* particleStates, unsigned int numParticles)
{
#define SQRT3INV 0.577350269 // = 1/sqrt{3}

    // directions to seed the new sub particles
    const float directions[] = {
                                    SQRT3INV,  SQRT3INV,  SQRT3INV, 
                                    SQRT3INV,  SQRT3INV, -SQRT3INV,
                                    SQRT3INV, -SQRT3INV,  SQRT3INV,
                                    SQRT3INV, -SQRT3INV, -SQRT3INV,
                                   -SQRT3INV,  SQRT3INV,  SQRT3INV,
                                   -SQRT3INV,  SQRT3INV, -SQRT3INV,
                                   -SQRT3INV, -SQRT3INV,  SQRT3INV,
                                   -SQRT3INV, -SQRT3INV, -SQRT3INV
                               };

    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    unsigned int id = particleIds[idx];
    char state = particleStates[id] & 15;

    // if parent particle makes transition from "default" -> "split" (3)
    // "default" -> "boundary" (2), "split" -> "boundary" (14) the sub particle needs
    // to be reinitialized 
     if (state == 2 || state == 3 || state == 14 || state == 11)
    {
        float density = particleSimulationData[id*SD_NUM_ELEMENTS + SD_DENSITY];
        float radicand = 3.0f*gSimParamsDev.particleMass/(4.0f*M_PI*density);
        float radius = pow(radicand, 1.0f/3.0f);
        float3 pos;
        pos.x = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_X];
        pos.y = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Y];
        pos.z = particleVertexData[id*VD_NUM_ELEMENTS + VD_POS_Z];

        // ... initialize initial position and velocity of the corresponding 
        // sub particles
        for (unsigned int i = 0; i < 8; i++) 
        {
            // update velocity
            int index = (8*id + i)*SD_NUM_ELEMENTS;
            subParticleSimulationData[index + SD_VEL0_X] = 
                particleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_X];
            subParticleSimulationData[index + SD_VEL0_Y] = 
                particleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_Y];
            subParticleSimulationData[index + SD_VEL0_Z] = 
                particleSimulationData[id*SD_NUM_ELEMENTS + SD_VEL0_Z];
           
            // update position
            index = (8*id + i)*VD_NUM_ELEMENTS;
            subParticleVertexData[index + VD_POS_X] = pos.x + 
                directions[3*i + 0]*radius;
            subParticleVertexData[index + VD_POS_Y] = pos.y + 
                directions[3*i + 1]*radius;
            subParticleVertexData[index + VD_POS_Z] = pos.z + 
                directions[3*i + 2]*radius;
        }
    }

#undef SQRT3INV
}
//-----------------------------------------------------------------------------
__global__ void check_split_boundary_default (char* particleState, 
        int* isSplit, int* isBoundary, int* isDefault)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    char state = particleState[idx] & 3;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    if (state == 0)
    {
        isDefault[idx] = 1;
    }
    else if (state == 2)
    {
        isBoundary[idx] = 1;
    }
    else
    {
        isSplit[idx] = 1;
    }
}
//-----------------------------------------------------------------------------
__global__ void collect_ids (int* subParticleIdList, 
    int* sortedSubParticleIdList,int* splitParticleIdList,
    int* boundaryParticleIdList, int* defaultParticleIdList,
    int* isSplit, int* isBoundary, int* isDefault, int* splitPrefixSum, 
    int* boundaryPrefixSum, int* defaultPrefixSum, 
    unsigned int numParticlesSplit)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= gSimParamsDev.numParticles)
    {
        return;
    }

    if (isSplit[idx] == 1)
    {
        int splitPreSum = splitPrefixSum[idx];

        for (unsigned int i = 0; i < 8; i++)
        {
            subParticleIdList[8*splitPreSum + i] = 8*idx + i;
            sortedSubParticleIdList[8*splitPreSum + i] = 8*idx + i;
        }

        splitParticleIdList[splitPreSum] = idx; 
    }
    else if (isBoundary[idx] == 1)
    {
        int boundaryPreSum = boundaryPrefixSum[idx];

        for (unsigned int i = 0; i < 8; i++)
        {
            subParticleIdList[8*(numParticlesSplit + boundaryPreSum) + i] =  
                8*idx + i;
            sortedSubParticleIdList[8*(numParticlesSplit + boundaryPreSum) 
                + i] = 8*idx + i;
        }

        boundaryParticleIdList[boundaryPreSum] = idx;
    }
    else
    {
        defaultParticleIdList[defaultPrefixSum[idx]] = idx;
    }
}
//__global__ void collision_handling(float* particleVertexData, 
//    float* particleSimulationData)
//{
//    /*unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
//
//    if (idx >= gSimParamsDev.numParticles)
//    {
//        return;
//    }
//
//    unsigned int idVert = idx*VD_NUM_ELEMENTS;
//    unsigned int idSim = idx*SD_NUM_ELEMENTS;
//
//    float3 pos;
//    float3 vel;
//
//    pos.x = tex1Dfetch(gParticleVertexData, idVert + VD_POS_X);
//    pos.y = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Y);
//    pos.z = tex1Dfetch(gParticleVertexData, idVert + VD_POS_Z);
//
//    vel.x = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_X);
//    vel.y = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Y);
//    vel.z = tex1Dfetch(gParticleSimulationData, idSim + SD_VEL0_Z);
//
//    //
//    unsigned int i,j,k;
//    i = (unsigned int)((pos.x - gBoundaryOrigin[0])/gDx);
//    j = (unsigned int)((pos.y - gBoundaryOrigin[1])/gDx);
//    k = (unsigned int)((pos.z - gBoundaryOrigin[2])/gDx);
//    unsigned int idx2 = i + gnBoundarySamples[0]*(j + gnBoundarySamples[1]*k);
//    unsigned int nodeIdx = tex1Dfetch(gIndexMap, idx2);
//    float dist = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_DISTANCE);
//    
//    float3 bNorm;
//
//    bNorm.x = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_X);
//    bNorm.y = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_Y);
//    bNorm.z = tex1Dfetch(gNodeTable, NC_NUM_ELEMENTS*nodeIdx + NC_NORMAL_Z);
//
//    if (bNorm.y != 0.0f)
//    {
//        particleVertexData[idVert + VD_POS_X] -= gSimParamsDev.timeStep*vel.x;
//        particleVertexData[idVert + VD_POS_Y] -= gSimParamsDev.timeStep*vel.y;
//        particleVertexData[idVert + VD_POS_Z] -= gSimParamsDev.timeStep*vel.z;
//    }*/
//}
//-----------------------------------------------------------------------------
__global__ void create_density_slice (float* data, unsigned int width, 
    unsigned int height, unsigned int depth)
{
    int u = blockIdx.x*blockDim.x + threadIdx.x;
    int v = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idx = width*v + u;

    if (u >= width || v >= height)
    {
        return;
    }

    float3 pos;
    pos.x = -1.0f + gSimParamsDev.gridSpacing*u;
    pos.y = gSimParamsDev.gridOrigin[1] + gSimParamsDev.gridSpacing*v;
    pos.z = gSimParamsDev.gridOrigin[2] + gSimParamsDev.gridSpacing*depth;

    float tu = (pos.x - gBoundaryGridOrigin[0])/gBoundaryGridLength[0];
    float tv = (pos.y - gBoundaryGridOrigin[1])/gBoundaryGridLength[1];
    float tw = (pos.z - gBoundaryGridOrigin[2])/gBoundaryGridLength[2];

    float density = tex3D(gBoundaryDistances, tu, tv, tw);

    data[idx] = density;
}
//-----------------------------------------------------------------------------
// definition of aux. functions (device) 
//-----------------------------------------------------------------------------
__device__ inline int3 compute_grid_coordinate(float3 pos, float d)
{
    int3 gridCoord;

    gridCoord.x = (unsigned int)((pos.x + d - gSimParamsDev.gridOrigin[0])/
        gSimParamsDev.gridSpacing);
    gridCoord.y = (unsigned int)((pos.y + d - gSimParamsDev.gridOrigin[1])/
        gSimParamsDev.gridSpacing);
    gridCoord.z = (unsigned int)((pos.z + d - gSimParamsDev.gridOrigin[2])/
        gSimParamsDev.gridSpacing);

    gridCoord.x = gridCoord.x%gSimParamsDev.gridDim[0];
    gridCoord.y = gridCoord.y%gSimParamsDev.gridDim[1];
    gridCoord.z = gridCoord.z%gSimParamsDev.gridDim[2];

    gridCoord.x = min(max(gridCoord.x, 0),gSimParamsDev.gridDim[0] - 1);
    gridCoord.y = min(max(gridCoord.y, 0),gSimParamsDev.gridDim[1] - 1);
    gridCoord.z = min(max(gridCoord.z, 0),gSimParamsDev.gridDim[2] - 1);

    return gridCoord;
}
//-----------------------------------------------------------------------------
__device__ inline int3 compute_grid_coordinate_sub_particles(float3 pos, 
    float d)
{
    int3 gridCoord;

    gridCoord.x = (unsigned int)((pos.x + d - gSimParamsDev.gridOrigin[0])/
        gSimParamsDev.gridSpacingSubParticles);
    gridCoord.y = (unsigned int)((pos.y + d - gSimParamsDev.gridOrigin[1])/
        gSimParamsDev.gridSpacingSubParticles);
    gridCoord.z = (unsigned int)((pos.z + d - gSimParamsDev.gridOrigin[2])/
        gSimParamsDev.gridSpacingSubParticles);

    gridCoord.x = gridCoord.x%gSimParamsDev.gridDimSubParticles[0];
    gridCoord.y = gridCoord.y%gSimParamsDev.gridDimSubParticles[1];
    gridCoord.z = gridCoord.z%gSimParamsDev.gridDimSubParticles[2];

    gridCoord.x = min(max(gridCoord.x, 0),
        gSimParamsDev.gridDimSubParticles[0] - 1);
    gridCoord.y = min(max(gridCoord.y, 0),
        gSimParamsDev.gridDimSubParticles[1] - 1);
    gridCoord.z = min(max(gridCoord.z, 0),
        gSimParamsDev.gridDimSubParticles[2] - 1);

    return gridCoord;
}
//-----------------------------------------------------------------------------
__device__ inline int compute_hash_from_grid_coordinate (int i, int j, int k)
{
    return gSimParamsDev.gridDim[0]*(gSimParamsDev.gridDim[1]*k + j) + i;
}
//-----------------------------------------------------------------------------
__device__ inline int compute_hash_from_grid_coordinate_sub_particle (int i, 
    int j, int k)
{
    return gSimParamsDev.gridDimSubParticles[0]*
        (gSimParamsDev.gridDimSubParticles[1]*k + j) + i;
}
//-----------------------------------------------------------------------------
__device__ inline float norm(const float3& a)
{
    return sqrt(a.x*a.x+a.y*a.y+a.z*a.z);
}
//-----------------------------------------------------------------------------
__device__ inline void normalize (float3& a)
{
    float norm = sqrt(a.x*a.x+a.y*a.y+a.z*a.z);
    a.x /= norm;
    a.y /= norm;
    a.z /= norm;
}
//-----------------------------------------------------------------------------
__device__ inline float compute_distance (float3 a, float3 b)
{
    return sqrt((a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y) 
        + (a.z-b.z)*(a.z-b.z));
}
//-----------------------------------------------------------------------------
__device__ inline float compute_squared_distance (float3 a, float3 b)
{
    return ((a.x-b.x)*(a.x-b.x) + (a.y-b.y)*(a.y-b.y) 
        + (a.z-b.z)*(a.z-b.z));
}
//-----------------------------------------------------------------------------
__device__ inline float dot_product (const float3& a, const float3& b)  
{
    return a.x*b.x + a.y*b.y + a.z*b.z;
}
//-----------------------------------------------------------------------------
__device__ inline void project_quantities_cell (float3& acc, float& density, 
    float& pressure, float& numNeighbors, const float3& xi, int start, int end)
{
    int j;
    float3 xj;  // neighbor particle's position
    float3 vj;  // neighbor particle's velocity
    float rhoj; // neighbor density
    float pj;   // neighbor pressure
    float3 aj;  // neighbor acceleration
    float h = gSimParamsDev.compactSupportSub;
    float sqDist, d;

    for (int i = start; i < end; i++)
    {
        j = tex1Dfetch(gSortedParticleIdList, i);
        
        xj.x = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS
            + VD_POS_X);
        xj.y = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS
            + VD_POS_Y);
        xj.z = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS
            + VD_POS_Z);
        rhoj = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_DENSITY);
        pj  = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_PRESSURE);
        aj.x = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_ACC_X);
        aj.y = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_ACC_Y);
        aj.z = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_ACC_Z);

        sqDist = compute_squared_distance(xi, xj);

        if (sqDist <= h*h)
        {
            density += rhoj;
            pressure += pj;
            acc.x += aj.x;
            acc.y += aj.y;
            acc.z += aj.z;
            numNeighbors += 1.0f;

            /*d = h*h - sqDist;
            density += gSimParamsDev.poly6Sub*rhoj;
            pressure += gSimParamsDev.poly6Sub*pj;
            velocity.x += gSimParamsDev.poly6Sub*vj.x*d*d*d;
            velocity.y += gSimParamsDev.poly6Sub*vj.y*d*d*d;
            velocity.z += gSimParamsDev.poly6Sub*vj.z*d*d*d;*/
        }

    }
}
//-----------------------------------------------------------------------------
//  Computes the contribution of neighborparticles of one particular grid cell
//  to the density of the particle at position [pos].
__device__ float compute_particle_density_cell (const float3 &pos, 
	float* particleVertexData, int* particleIdList, int start, int end)
{
    int particleIndex; // index of the neighbor of the particle
    float density = 0.0f;
    float3 p; // neighbor particle's position
    float h = gSimParamsDev.compactSupport;
    float r;
    float d;

    for (int i = start; i < end; i++) 
    {
        particleIndex = particleIdList[i];

        // compute position of the neighbor
        p.x = particleVertexData[particleIndex*VD_NUM_ELEMENTS
            + VD_POS_X];
        p.y = particleVertexData[particleIndex*VD_NUM_ELEMENTS
            + VD_POS_Y];
        p.z = particleVertexData[particleIndex*VD_NUM_ELEMENTS
            + VD_POS_Z];

        r = compute_distance(p, pos);
        
        // TODO: evaluating r*r <= h*h might save taking the sqrt in 
        // compute_distance proc. 
        if (r <= h) 
        {
            d = h*h - r*r;
            density += gSimParamsDev.poly6*d*d*d;
        }
    }

    return density;
}
//-----------------------------------------------------------------------------
//  Computes the contribution of neighborsub particles of one particular grid 
//  cell to the density of the particle at position [pos].
__device__ float compute_sub_particle_density_cell (const float3 &pos, 
	float* particleVertexData, int* particleIdList, int start, int end)
{
    int particleIndex; // index of the neighbor of the particle
    float density = 0.0f;
    float3 p; // neighbor particle's position
    float h = gSimParamsDev.compactSupportSub;
    float r;
    float d;

    for (int i = start; i < end; i++) 
    {
        particleIndex = particleIdList[i];

        // compute position of the neighbor
        p.x = particleVertexData[particleIndex*VD_NUM_ELEMENTS
            + VD_POS_X];
        p.y = particleVertexData[particleIndex*VD_NUM_ELEMENTS
            + VD_POS_Y];
        p.z = particleVertexData[particleIndex*VD_NUM_ELEMENTS
            + VD_POS_Z];

        r = compute_distance(p, pos);
        
        // TODO: evaluating r*r <= h*h might save taking the sqrt in 
        // compute_distance proc. 
        if (r <= h) 
        {
            d = h*h - r*r;
            density += gSimParamsDev.poly6Sub*d*d*d;
        }
    }

    return density;
}
//-----------------------------------------------------------------------------
__device__ inline void compute_viscosity_pressure_forces_and_ifsurf_cell
    (const float3& xi, float rhoi, float pi, const float3& vi,
    float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int start, int end, 
    float3* pressureForce, float3* viscosityForce, float3* colGra, 
    float* colLapl, float3* sumPosNeighbor, float* nNeighbors)
{
    int j;      // neighbor index in particle list
    float3 xj;  // neighbor particle's position
    float3 vj;  // neighbor particle's velocity
    float rhoj; // neighbor density
    float pj;   // neighbor pressure
    float3 r;   // xi - xj
    float rn;   // ||xi - xj||
    float h = gSimParamsDev.compactSupport; // effective radius
    float grad  = gSimParamsDev.gradSpiky;
    float lapl  = gSimParamsDev.laplVisc;
    float grad2 = gSimParamsDev.gradPoly6;
    float lapl2 = gSimParamsDev.laplPoly6;

    float pressure; // pressure term in the kernel approx
    float rhoi2 = rhoi*rhoi;                    
    float m = gSimParamsDev.particleMass;
    float mu = gSimParamsDev.dynamicViscosity;

    float d; // helper value to avoid arithmetic operations

    for (int i = start; i < end; i++) 
    {
        // get neighbor index from particle list
        j = tex1Dfetch(gSortedParticleIdList, i); 

        // get neighbor particle information
        xj.x = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS + VD_POS_X);
        xj.y = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS + VD_POS_Y);
        xj.z = tex1Dfetch(gParticleVertexData, j*VD_NUM_ELEMENTS + VD_POS_Z);
        vj.x = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_VEL0_X);
        vj.y = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_VEL0_Y);
        vj.z = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_VEL0_Z);
        rhoj = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_DENSITY);
        pj   = tex1Dfetch(gParticleSimulationData, j*SD_NUM_ELEMENTS
            + SD_PRESSURE);

        r.x = xi.x - xj.x;
        r.y = xi.y - xj.y;
        r.z = xi.z - xj.z;

        rn = norm(r);
        
        // TODO: * masse koennte ausgeklammert werden um multiplikationen
        //         zu sparen.
        //       * generell kann der pressure term in hinblick auf rhoi und
        //         pi vereinfacht werden.
        //       * visc force: mu koennte ausgeklammert werden etc.
        //       * zwei float3's fuer beide kraefte koennten genutzt werden
        //         um die terme zu vereinfachen.
        pressure = rhoi*(pi/rhoi2 + pj/(rhoj*rhoj))*m;

        if (rn <= h && rn > 0.0f)
        {
            // compute pressure force
            d = (h-rn)*(h-rn);

            pressureForce->x -= pressure*grad*d/rn*r.x;
            pressureForce->y -= pressure*grad*d/rn*r.y;
            pressureForce->z -= pressure*grad*d/rn*r.z;
        
            // compute viscosity force
            d = (h-rn);

            viscosityForce->x += mu*(vj.x-vi.x)*m/rhoj*lapl*d;
            viscosityForce->y += mu*(vj.y-vi.y)*m/rhoj*lapl*d;
            viscosityForce->z += mu*(vj.z-vi.z)*m/rhoj*lapl*d;

            // compute color gradient
            d = (h*h-rn*rn)*(h*h-rn*rn);

            colGra->x += m/rhoj*grad2*d*r.x;
            colGra->y += m/rhoj*grad2*d*r.y;
            colGra->z += m/rhoj*grad2*d*r.z;

            // compute color laplacian
            d = (h*h - rn*rn)*(3.0f*h*h - 7.0f*rn*rn);

            *colLapl += m/rhoj*lapl2*d;

            //
            sumPosNeighbor->x += xj.x;
            sumPosNeighbor->y += xj.y;
            sumPosNeighbor->z += xj.z;
            *nNeighbors += 1.0f;
        }
    }
}
//-----------------------------------------------------------------------------
__device__ inline void compute_sub_particle_viscosity_pressure_forces_cell
    (const float3& xi, float rhoi, float pi, const float3& vi,
    float* particleVertexData, float* particleSimulationData, 
    int* particleIdList, int start, int end, 
    float3* force, float3* colGra, float* colLapl)
{
    int j;      // neighbor index in particle list
    float3 xj;  // neighbor particle's position
    float3 vj;  // neighbor particle's velocity
    float rhoj; // neighbor density
    float pj;   // neighbor pressure
    float3 r;   // xi - xj
    float rn;   // ||xi - xj||
    float h = gSimParamsDev.compactSupportSub; // effective radius
    float grad  = gSimParamsDev.gradSpikySub;
    float lapl  = gSimParamsDev.laplViscSub;
    float grad2 = gSimParamsDev.gradPoly6Sub;
    float lapl2 = gSimParamsDev.laplPoly6Sub;

    float pressure; // pressure term in the kernel approx
    float rhoi2 = rhoi*rhoi;                    
    float m = gSimParamsDev.subParticleMass;
    float mu = gSimParamsDev.dynamicViscosity;

    float d; // helper value to avoid arithmetic operations

    for (int i = start; i < end; i++) 
    {
        // get neighbor index from particle list
        j = particleIdList[i]; 

        // get neighbor particle information
        xj.x = particleVertexData[j*VD_NUM_ELEMENTS + VD_POS_X];
        xj.y = particleVertexData[j*VD_NUM_ELEMENTS + VD_POS_Y];
        xj.z = particleVertexData[j*VD_NUM_ELEMENTS + VD_POS_Z];
        vj.x = particleSimulationData[j*SD_NUM_ELEMENTS + SD_VEL0_X];
        vj.y = particleSimulationData[j*SD_NUM_ELEMENTS + SD_VEL0_Y];
        vj.z = particleSimulationData[j*SD_NUM_ELEMENTS + SD_VEL0_Z];
        rhoj = particleSimulationData[j*SD_NUM_ELEMENTS + SD_DENSITY];
        pj   = particleSimulationData[j*SD_NUM_ELEMENTS + SD_PRESSURE];

        r.x = xi.x - xj.x;
        r.y = xi.y - xj.y;
        r.z = xi.z - xj.z;

        rn = norm(r);
        
        // TODO: * masse koennte ausgeklammert werden um multiplikationen
        //         zu sparen.
        //       * generell kann der pressure term in hinblick auf rhoi und
        //         pi vereinfacht werden.
        //       * visc force: mu koennte ausgeklammert werden etc.
        //       * zwei float3's fuer beide kraefte koennten genutzt werden
        //         um die terme zu vereinfachen.
        pressure = rhoi*(pi/rhoi2 + pj/(rhoj*rhoj))*m;

        if (rn <= h && rn > 0.0f)
        {
            // compute pressure force
            d = (h-rn)*(h-rn);

            force->x -= pressure*grad*d/rn*r.x;
            force->y -= pressure*grad*d/rn*r.y;
            force->z -= pressure*grad*d/rn*r.z;
        
            // compute viscosity force
            d = (h - rn);

            force->x += mu*(vj.x - vi.x)*m/rhoj*lapl*d;
            force->y += mu*(vj.y - vi.y)*m/rhoj*lapl*d;
            force->z += mu*(vj.z - vi.z)*m/rhoj*lapl*d;

            // compute color gradient
            d = (h*h - rn*rn)*(h*h - rn*rn);

            colGra->x += m/rhoj*grad2*d*r.x;
            colGra->y += m/rhoj*grad2*d*r.y;
            colGra->z += m/rhoj*grad2*d*r.z;

            // compute color laplacian
            d = (h*h - rn*rn)*(3.0f*h*h - 7.0f*rn*rn);

            *colLapl += m/rhoj*lapl2*d;
        }
    }
}
//-----------------------------------------------------------------------------
template<typename T>
__global__ void copy_array (T* dst, T* src, unsigned int numElements)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= numElements)
    {
        return;
    }

    dst[idx] = src[idx];
}
//-----------------------------------------------------------------------------
//  HOST CODE
//-----------------------------------------------------------------------------

#define EMPTY_CELL 0xFFFFFFFF

//-----------------------------------------------------------------------------
//  forward declaration of aux. functions
//-----------------------------------------------------------------------------
void create_particle_box (float sx, float sy, float sz, float d, 
    unsigned int numParticles, float** particleVD, float** particleSD,
    unsigned int* numParticlesCreated);
void set_simulation_domain (float xs, float ys, float zs, float xe,
    float ye, float ze, float gridSpacing, float gridSpacingSubParticles,
    SimulationParameters* parameters);
void compute_particle_kernel_invocation_information 
    (unsigned int& nThreadsBlock, unsigned int& nBlocks, 
    unsigned int numParticles);
void set_up_3d_float_texture (struct textureReference* texref, 
    hipArray* arr, float* data, unsigned int dim[3]);

//-----------------------------------------------------------------------------
//  Definition of ParticleSimulation class 
//-----------------------------------------------------------------------------
ParticleSimulation::ParticleSimulation (): mParticleVertexData(NULL), 
    mParticleSimulationData(NULL), mParticleVertexDataDevPtr(NULL),
    mParticleSimulationDataDevPtr(NULL), mParticleIdsDevPtr(NULL),
    mParticleHashListDevPtr(NULL), mCellStartListDevPtr(NULL), 
    mCellEndListDevPtr(NULL), mIsSurfaceParticleDevPtr(NULL), mParticleVertexDataVbo(0),
    mNumBlocks(0), mThreadsPerBlock(0), mNumSubParticles(0), mNumTimeSteps(0)
{
    memset(&mParameters, 0, sizeof(SimulationParameters));
}
//-----------------------------------------------------------------------------
ParticleSimulation::~ParticleSimulation() 
{
    // free host memory
    saveDeleteArray<float>(&mParticleVertexData);
    saveDeleteArray<float>(&mParticleSimulationData);
    
    // free device memory

    // free cuda memory
    cudaSafeFree<float>(&mParticleVertexDataDevPtr);
    cudaSafeFree<float>(&mParticleSimulationDataDevPtr);
    cudaSafeFree<float>(&mSubParticleVertexDataDevPtr);
    cudaSafeFree<float>(&mSubParticleSimulationDataDevPtr);
    cudaSafeFree<char>(&mParticleStatesDevPtr);
    cudaSafeFree<int>(&mParticleHashListDevPtr);
    cudaSafeFree<int>(&mCellStartListDevPtr);
    cudaSafeFree<int>(&mCellEndListDevPtr);

    cudaSafeFree<int>(&mIsSurfaceParticleDevPtr);
    cudaSafeFree<int>(&mParticleIdsDevPtr);
    cudaSafeFree<int>(&_isSplitDevPtr);     
    cudaSafeFree<int>(&_isBoundaryDevPtr);
    cudaSafeFree<int>(&_isDefaultDevPtr);
    cudaSafeFree<int>(&_splitPrefixSumDevPtr);
    cudaSafeFree<int>(&_boundaryPrefixSumDevPtr);
    cudaSafeFree<int>(&_defaultPrefixSumDevPtr);
    cudaSafeFree<int>(&mParticleIdsSplitDevPtr);
    cudaSafeFree<int>(&mParticleIdsDefaultDevPtr);
    cudaSafeFree<int>(&mSubParticleIdsDevPtr);
    cudaSafeFree<int>(&mParticleIdsBoundaryDevPtr);     
    cudaSafeFree<int>(&mParticleIdsSplitDevPtr);
    cudaSafeFree<int>(&mSubParticleHashsDevPtr);

    cudaSafeFree<int>(&mSubParticleCellStartIdsDevPtr);
    cudaSafeFree<int>(&mSubParticleCellEndIdsDevPtr);
 
    
    
    
    
    // free OpenGL vertex buffer object
    if (mParticleVertexDataVbo != 0) 
    {
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[0]) );
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[1]) );
        //cudaGLUnregisterBufferObject(mParticleVertexDataVbo); // <- deprecated
        glDeleteBuffers(1, &mParticleVertexDataVbo);
        glDeleteBuffers(1, &mSubParticleVertexDataVbo);
        mParticleVertexDataVbo = 0;
        mSubParticleVertexDataVbo = 0;
    }

}
//-----------------------------------------------------------------------------
ParticleSimulation* ParticleSimulation::Example01 () 
{
    // create a particle simulation 
    ParticleSimulation* sim = new ParticleSimulation();

    // create box (cube) of particles
    create_particle_box(-0.45f, -0.25f, -0.25f, 0.5f, 40000, 
        &sim->mParticleVertexData, &sim->mParticleSimulationData,
        &sim->mParameters.numParticles);

    if (sim->mParticleVertexData == NULL || 
        sim->mParticleSimulationData == NULL) 
    {
        THROW_EXCEPTION("Could not allocate memory for particles (Host).");
    }

    // set sph simulation related parameters
    sim->mParameters.kernelParticles = 20;
    sim->mParameters.restDensity = 998.648f;
    sim->mParameters.particleMass = sim->mParameters.restDensity*0.5f*0.5f*0.5f/
        static_cast<float>(sim->mParameters.numParticles);
    sim->mParameters.subParticleMass = sim->mParameters.particleMass/8.0f;
    sim->mParameters.gasStiffness = 3.0f;
    sim->mParameters.dynamicViscosity = 3.0f;
    sim->mParameters.gravity = 9.81f;
    sim->mParameters.tensionCoefficient = 0.0728f;
    sim->mParameters.normThresh = 15.065f;

    // compute the kernel radius
    float h = powf((3.0f*0.5f*0.5f*0.5f*sim->mParameters.kernelParticles)/
        (4.0f*M_PI*sim->mParameters.numParticles), 1.0f/3.0f);

    sim->mParameters.compactSupport =  h;
    sim->mParameters.compactSupportSub = h/2.0f;
    sim->mParameters.poly6 =  315.0f/(64.0f*M_PI*h*h*h*h*h*h*h*h*h);
    sim->mParameters.gradPoly6 = -945.0f/(32.0f*M_PI*h*h*h*h*h*h*h*h*h);
    sim->mParameters.laplPoly6 = -945.0f/(32.0f*M_PI*h*h*h*h*h*h*h*h*h);
    sim->mParameters.gradSpiky = -45.0f/(M_PI*h*h*h*h*h*h);
    sim->mParameters.laplVisc  =  45.0f/(M_PI*h*h*h*h*h*h);
    sim->mParameters.poly6Sub  =  sim->mParameters.poly6*512.0f;
    sim->mParameters.gradPoly6Sub = sim->mParameters.gradPoly6*512.0f;
    sim->mParameters.laplPoly6Sub = sim->mParameters.laplPoly6*512.0f;
    sim->mParameters.gradSpikySub = sim->mParameters.gradSpiky*64.0f;
    sim->mParameters.laplViscSub  =  sim->mParameters.laplVisc*64.0f;
    sim->mParameters.
    
    timeStep  = 0.003;
    sim->mParameters.timeStepSubParticles = 0.001f;
    
    set_simulation_domain(-1.0f, -1.0f, -1.0f, 1.0f, 1.0f, 1.0f, h, h/2.0f,
        &sim->mParameters);

    // set fluid volume
    sim->mParameters.fluidVolume = 0.5f*0.5f*0.5f; 

    // set parameters for boundary handling
    sim->mParameters.restitution = 0.0f;
    sim->mParameters.boxCen[0] = 0.0f;
    sim->mParameters.boxCen[1] = 0.0f;
    sim->mParameters.boxCen[2] = 0.0f;
    sim->mParameters.boxDim[0] = 0.7f;    
    sim->mParameters.boxDim[1] = 0.5f;    
    sim->mParameters.boxDim[2] = 0.3f;    

    // set parameters for new boundary handling
    sim->_boundaryMapFileName = std::string("icosphere.txt");

    // set parameters for surface extraction
    sim->mParameters.cmDistanceThresh = 0.5f;
    sim->mParameters.nPartTresh = 20.0f;
    sim->_leftI = 0.0f;
    sim->_rightI = 1.0f;


    //printf("h %")
    return sim;
}
//-----------------------------------------------------------------------------
int* ParticleSimulation::CreateIsParticleSurfaceList
    (const ParticleSimulation* sim)
{
    int* isSurfaceParticleList = new int[sim->mParameters.numParticles];
    
    CUDA_SAFE_CALL( hipMemcpy(isSurfaceParticleList, 
        sim->mIsSurfaceParticleDevPtr,
        sizeof(int)*sim->mParameters.numParticles, 
        hipMemcpyDeviceToHost) );

    int extr = 0;
    for (unsigned int i = 0; i < sim->mParameters.numParticles; i++) 
    {
       extr += isSurfaceParticleList[i];
    }

    printf("%d of %d extracted\n", extr, sim->mParameters.numParticles);
    

    return isSurfaceParticleList;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::FreeIsParticleSurfaceList 
    (int** isSurfaceParticleList)
{
    if (*isSurfaceParticleList == NULL) 
    {
        return;
    }

    delete[] *isSurfaceParticleList;
    *isSurfaceParticleList = NULL;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::freeAll () 
{
    // free host memory
    saveDeleteArray<float>(&mParticleVertexData);
    saveDeleteArray<float>(&mParticleSimulationData);
    
    // free device memory

    // free cuda memory
    cudaSafeFree<float>(&mParticleSimulationDataDevPtr);
    cudaSafeFree<int>(&mParticleIdsDevPtr);
    cudaSafeFree<int>(&mParticleHashListDevPtr);
    cudaSafeFree<int>(&mCellStartListDevPtr);
    cudaSafeFree<int>(&mCellEndListDevPtr);
    cudaSafeFree<int>(&mIsSurfaceParticleDevPtr);
    
    // free OpenGL vertex buffer object
    if (mParticleVertexDataVbo != 0) 
    {
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[0]) );
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[1]) );
        //cudaGLUnregisterBufferObject(mParticleVertexDataVbo); // <- deprecated
        glDeleteBuffers(1, &mParticleVertexDataVbo);
        glDeleteBuffers(1, &mSubParticleVertexDataVbo);
        mParticleVertexDataVbo = 0;
        mSubParticleVertexDataVbo = 0;
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::Init () 
{
    //
    // free device memory, if previously allocated 
    //

    // free cuda memory
    cudaSafeFree<float>(&mParticleSimulationDataDevPtr);
    cudaSafeFree<int>(&mParticleIdsDevPtr);
    cudaSafeFree<int>(&mParticleHashListDevPtr);
    cudaSafeFree<int>(&mCellStartListDevPtr);
    cudaSafeFree<int>(&mCellEndListDevPtr);
    cudaSafeFree<int>(&mIsSurfaceParticleDevPtr);
    
    // free OpenGL vertex buffer object
    if (mParticleVertexDataVbo != 0) 
    {
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[0]) );
        CUDA_SAFE_CALL( hipGraphicsUnregisterResource(mGraphicsResources[1]) );
        glDeleteBuffers(1, &mParticleVertexDataVbo);
        glDeleteBuffers(1, &mSubParticleVertexDataVbo);
        mParticleVertexDataVbo = 0;
        mSubParticleVertexDataVbo = 0;
    }

    //
    // allocate cuda device memory for storing the particles' vertex and
    // simulation data.
    // Vertex data is allocated on device using OpenGL, as it is stored
    // in an vertex buffer object, which is used for rendering later.
    //

    // Simulation data is allocated through cuda.
    CUDA_SAFE_CALL( hipMalloc(&mParticleSimulationDataDevPtr, 
        mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS) );

    // copy initial host data to device
    CUDA_SAFE_CALL( hipMemcpy(mParticleSimulationDataDevPtr, 
        mParticleSimulationData, 
        mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS,
        hipMemcpyHostToDevice) );
    
    // Vertex data is allocated through a vertex buffer object
    // the vbo is then registered to be used with CUDA
    glGenBuffers(1, &mParticleVertexDataVbo);
    glBindBuffer(GL_ARRAY_BUFFER, mParticleVertexDataVbo);
    glBufferData(GL_ARRAY_BUFFER, 
        mParameters.numParticles*VD_NUM_ELEMENTS*sizeof(float),
          mParticleVertexData, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[0], 
        mParticleVertexDataVbo, cudaGraphicsMapFlagsNone) );
    //cudaGLRegisterBufferObject(mParticleVertexDataVbo); // <- is deprecated
    
    //
    // alloc & Init additional aux. arrays for nearest neighbor search
    //
    const int* dim = mParameters.gridDim; 
    unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(int);

    CUDA_SAFE_CALL( hipMalloc(&mCellStartListDevPtr, size) );
    CUDA_SAFE_CALL( hipMalloc(&mCellEndListDevPtr, size) );

    // set each cell to be empty
    CUDA_SAFE_CALL( hipMemset(mCellStartListDevPtr, EMPTY_CELL, size) );
    CUDA_SAFE_CALL( hipMemset(mCellEndListDevPtr, EMPTY_CELL, size) );
     
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleHashListDevPtr, 
        mParameters.numParticles*sizeof(int)) );

    // alloc dev memory for surface particle extraction
    CUDA_SAFE_CALL( hipMalloc(&mIsSurfaceParticleDevPtr, 
        mParameters.numParticles*sizeof(int)) );


    try
    {
        this->allocateMemoryTwoScale();
    }
    catch (std::runtime_error& e)
    {
        std::cout << e.what() << std::endl;        
        system("pause");
    }


    // set up textures, for faster memory look-ups through caching
    // NOTE: VertexData needs to be mapped to get a valid device pointer, 
    //       as it is initial not allocated through CUDA's malloc
    hipChannelFormatDesc descf = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat);
    hipChannelFormatDesc desci = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindSigned);
    hipChannelFormatDesc descu = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindUnsigned);

    CUDA_SAFE_CALL ( hipBindTexture(0, gParticleSimulationData, 
        mParticleSimulationDataDevPtr, descf, 
        sizeof(float)*SD_NUM_ELEMENTS*mParameters.numParticles) );
    this->map();
    CUDA_SAFE_CALL ( hipBindTexture(0, gParticleVertexData, 
        mParticleVertexDataDevPtr, descf, 
        sizeof(float)*VD_NUM_ELEMENTS*mParameters.numParticles) );
    this->unmap();
    CUDA_SAFE_CALL ( hipBindTexture(0, gCellStartList, mCellStartListDevPtr, 
        desci, size) );
    CUDA_SAFE_CALL ( hipBindTexture(0, gCellEndList, mCellEndListDevPtr, 
        desci, size) );
    CUDA_SAFE_CALL ( hipBindTexture(0, gSortedParticleIdList, mParticleIdsDevPtr, 
        desci, mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL ( hipBindTexture(0, gParticleHashList, mParticleHashListDevPtr, 
        desci, mParameters.numParticles*sizeof(int)) );

    // set number of CUDA blocks and threads per blocks for each kernel 
    // invocation
    // NOTE:  - chose different values than 256 to try to get more performance
    //        - make threadsPerBlock and blocks function parameters
    compute_particle_kernel_invocation_information(mThreadsPerBlock, mNumBlocks, 
        mParameters.numParticles);

    this->setUpSphInComplexShapes();
    
}
//-----------------------------------------------------------------------------
// allocates and initializes memory needed for the two scale particle 
// simulation
void ParticleSimulation::allocateMemoryTwoScale ()
{
    // create opengl vbo for storing the vertex information of the 
    // sub particles
    glGenBuffers(1, &mSubParticleVertexDataVbo);
    glBindBuffer(GL_ARRAY_BUFFER, mSubParticleVertexDataVbo);
    glBufferData(GL_ARRAY_BUFFER, 8*mParameters.numParticles*VD_NUM_ELEMENTS*
        sizeof(float), NULL, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[1], 
        mSubParticleVertexDataVbo, cudaGraphicsMapFlagsNone) );

    // create opengl vbo for storing the ids of the particles in 
    // default state
    glGenBuffers(1, &mParticleIdsDefaultVbo);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mParticleIdsDefaultVbo);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, mParameters.numParticles*sizeof(int),
        NULL, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[2], 
        mParticleIdsDefaultVbo, cudaGraphicsMapFlagsNone) );

    // create opengl vbo for storing the ids of the active sub particles 
    glGenBuffers(1, &mSubParticleIdsVbo);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mSubParticleIdsVbo);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, mParameters.numParticles*sizeof(int)*8,
        NULL, GL_DYNAMIC_COPY);
    CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&mGraphicsResources[3], 
        mSubParticleIdsVbo, cudaGraphicsMapFlagsNone) );

    CUDA_SAFE_CALL( hipMalloc(&mSubParticleSortedIdsDevPtr, 
        sizeof(int)*8*mParameters.numParticles) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleSimulationDataDevPtr, 
        8*mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS) );
    CUDA_SAFE_CALL( hipMemset(mSubParticleSimulationDataDevPtr, 0,
        8*mParameters.numParticles*sizeof(float)*SD_NUM_ELEMENTS) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleHashsDevPtr, 
        8*mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleStatesDevPtr, 
               mParameters.numParticles*sizeof(char)) );
    CUDA_SAFE_CALL( hipMemset(mParticleStatesDevPtr, 0, 
        mParameters.numParticles*sizeof(char)) );
    mParticleStates = new char[mParameters.numParticles];    
    CUDA_SAFE_CALL( hipMalloc(&_isSplitDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&_isBoundaryDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&_isDefaultDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&_splitPrefixSumDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );    
    CUDA_SAFE_CALL( hipMalloc(&_boundaryPrefixSumDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );    
    CUDA_SAFE_CALL( hipMalloc(&_defaultPrefixSumDevPtr, 
        (mParameters.numParticles + 1)*sizeof(int)) );    
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsDefaultDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsBoundaryDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mParticleIdsSplitDevPtr, 
        mParameters.numParticles*sizeof(int)) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleCellStartIdsDevPtr, 
        sizeof(int)*mParameters.gridDimSubParticles[0]*
        mParameters.gridDimSubParticles[1]*
        mParameters.gridDimSubParticles[2]) );
    CUDA_SAFE_CALL( hipMalloc(&mSubParticleCellEndIdsDevPtr, 
        sizeof(int)*mParameters.gridDimSubParticles[0]*
        mParameters.gridDimSubParticles[1]*
        mParameters.gridDimSubParticles[2]) );
}
//-----------------------------------------------------------------------------
void ParticleSimulation::Bind () const 
{
    // copy simulation parameters to constant memory on device.
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gSimParamsDev), (void*)&mParameters, 
        sizeof(SimulationParameters)) );  
}
//-----------------------------------------------------------------------------
void ParticleSimulation::Advance ()
{    
    try
    {
        this->map();
        this->computeParticleHash();
        this->sortParticleIdsByHash();
        this->computeCellStartEndList();
        this->computeDensityPressure();
        this->computeAcceleration();
        //this->computeParticleState();
        ///this->collect();
        //this->initializeSubParticles(); 
        //this->computeSubParticleHash();
        //this->sortSubParticleIdsByHash();
        //this->computeSubParticleCellStartEndList();
        //this->projectQuantities();
        //this->computeSubParticleDensityPressure(); 
        //this->computeSubParticleAcceleration();
        this->integrate();
        //this->integrateSubParticles();
        //this->handleCollisions();
        //this->handleSubParticleCollisions();
        this->unmap();

        mNumTimeSteps++;
    }
    catch (runtime_error& e)
    {  
        cout << e.what() << endl;
        system("pause");
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::AdvanceSubParticles ()
{
    try
    {
        this->map();
        this->computeSubParticleHash();
        this->sortSubParticleIdsByHash();
        this->computeSubParticleCellStartEndList();
        this->computeSubParticleDensityPressure(); 

        this->projectQuantities();
        mTimer.Start();
        this->computeSubParticleAcceleration();
        mTimer.Stop();
        mTimer.DumpElapsed();
        this->integrateSubParticles();
        this->handleSubParticleCollisions();
        this->unmap();
    }
    catch (runtime_error& e)
    {
        cout << e.what() << endl;
        system("pause");
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::AdvanceTwoScale ()
{
    try
    {
        this->map();

        this->computeParticleHash();
        this->sortParticleIdsByHash();
        this->computeCellStartEndList();
        this->computeDensityPressure();
        this->computeAcceleration();
        
        this->computeParticleState();
        this->collect();
        this->initializeSubParticles(); 
        this->projectQuantities();

        this->computeSubParticleHash();
        this->sortSubParticleIdsByHash();
        this->computeSubParticleCellStartEndList();
        this->computeSubParticleDensityPressure(); 
        this->computeSubParticleAcceleration();
    
        this->integrate();
        this->integrateSubParticles();
        this->handleCollisions();
        this->handleSubParticleCollisions();

        this->unmap();

        mNumTimeSteps++;
    }
    catch (runtime_error& e)
    {  
        cout << e.what() << endl;
        system("pause");
    }    
}
//-----------------------------------------------------------------------------
void ParticleSimulation::Check3DTextures () const
{
    // compute a higher res slice of the density data using intrinsic trilinear
    // interpolation to check of the textures have been set up correctly.

    unsigned int width = mParameters.gridDim[0];
    unsigned int height = mParameters.gridDim[1];
    float* sliceDataDevPtr;

    CUDA_SAFE_CALL( hipMalloc(&sliceDataDevPtr, sizeof(float)*width*height) );
    
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(width/blockSize.x + 1, height/blockSize.y + 1);
    
    create_density_slice <<<gridSize, blockSize>>> (sliceDataDevPtr, 
        width, height, mParameters.gridDim[2]/2);

    float* sliceData = new float[width*height];
    CUDA_SAFE_CALL( hipMemcpy(sliceData, sliceDataDevPtr, 
        sizeof(float)*width*height, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipFree(sliceDataDevPtr) );
    
    PortablePixmap ppm(width, height, 255);

    float maxDensity = mBoundaryHandling->ComputeMaxDensity();
    float restDistance = mBoundaryHandling->GetRestDistance();
    
    for (unsigned int j = 0; j < height; j++)
    {
        for (unsigned int i = 0; i < width; i++)
        {
            unsigned int idx = i + j*width;
            float density = sliceData[idx];
           
            ppm.setJET(i,j, std::abs(density)/restDistance);
        }
    }

    ppm.save("3dtextest.ppm");

    delete[] sliceData;
}
//-----------------------------------------------------------------------------
float ParticleSimulation::GetParticleRadius () const
{
    return powf((3.0*mParameters.fluidVolume)/
        (4.0*M_PI*mParameters.numParticles), 1.0f/3.0f);
}
//-----------------------------------------------------------------------------
float ParticleSimulation::GetSubParticleRadius () const
{
    return 0.5f*this->GetParticleRadius();
}
//-----------------------------------------------------------------------------
const char* ParticleSimulation::GetParticleState () const
{
    CUDA_SAFE_CALL( hipMemcpy(mParticleStates, mParticleStatesDevPtr, 
        sizeof(char)*mParameters.numParticles, hipMemcpyDeviceToHost) );
    return mParticleStates;
}
//-----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumParticles () const
{
    return mParameters.numParticles;
}
//-----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumTimesSteps () const
{
    return mNumTimeSteps;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::SetNPartThresh (float dVal)
{
    mParameters.nPartTresh += dVal;
    printf("# particle thresh %f\n", mParameters.nPartTresh);
    this->Bind();
}
//-----------------------------------------------------------------------------
void ParticleSimulation::DecreaseCmDistanceThresh ()
{
    _rightI = mParameters.cmDistanceThresh;
    mParameters.cmDistanceThresh = 0.5f*(_rightI - _leftI);
    printf("cmDistance = %f\n", mParameters.cmDistanceThresh);
    this->Bind();
}
//-----------------------------------------------------------------------------
void ParticleSimulation::IncreaseCmDistanceThresh ()
{
    _leftI = mParameters.cmDistanceThresh;
    mParameters.cmDistanceThresh = 0.5f*(_rightI - _leftI);
    printf("cmDistance = %f\n", mParameters.cmDistanceThresh);
    this->Bind();
}
//-----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLParticleVertexBufferObject () const
{
    return mParticleVertexDataVbo;
}
//-----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLParticleIndexVertexBufferObject () const
{
    return mParticleIdsDefaultVbo;
}
//-----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumParticlesDefault () const
{
    return mNumParticlesDefault;
}
//----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLSubParticleVertexBufferObject () const
{
    return mSubParticleVertexDataVbo;
}
//----------------------------------------------------------------------------
GLuint ParticleSimulation::GetGLSubParticleIndexVertexBufferObject () const
{
    return mSubParticleIdsVbo;
}
//----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumSubParticles () const
{
    return mNumSubParticles;
}
//----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumSubParticlesRegular () const
{
    return mNumParticlesSplit*8;
}
//----------------------------------------------------------------------------
unsigned int ParticleSimulation::GetNumSubParticlesBoundary () const
{
    return mNumParticlesBoundary*8;
}
//-----------------------------------------------------------------------------
// Definition of private methods
//-----------------------------------------------------------------------------
void ParticleSimulation::computeParticleHash () 
{
    compute_particle_hash <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleIdsDevPtr, 
        mParticleHashListDevPtr, mParameters.numParticles);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeSubParticleHash ()
{
    if (mNumSubParticles != 0)
    {
        compute_sub_particle_hash
            <<< mNumBlocksSubParticle, mThreadsPerBlockSubParticle >>> 
            (mSubParticleVertexDataDevPtr, mSubParticleIdsDevPtr, 
            mSubParticleHashsDevPtr, mNumSubParticles);
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::sortParticleIdsByHash ()
{
   thrust::sort_by_key(thrust::device_ptr<int>(mParticleHashListDevPtr),
        thrust::device_ptr<int>(mParticleHashListDevPtr + 
        mParameters.numParticles), 
        thrust::device_ptr<int>(mParticleIdsDevPtr));
}
//-----------------------------------------------------------------------------
void ParticleSimulation::sortSubParticleIdsByHash ()
{
    copy_array <int> <<< mNumBlocksSubParticle, mThreadsPerBlockSubParticle >>>
        (mSubParticleSortedIdsDevPtr, mSubParticleIdsDevPtr, mNumSubParticles);
    thrust::sort_by_key(thrust::device_ptr<int>(mSubParticleHashsDevPtr),
        thrust::device_ptr<int>(mSubParticleHashsDevPtr + 
        mNumSubParticles),
        thrust::device_ptr<int>(mSubParticleSortedIdsDevPtr));
}

//-----------------------------------------------------------------------------
void ParticleSimulation::computeCellStartEndList () 
{
    int* dim = mParameters.gridDim; 
    unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(int);

    hipMemset(mCellStartListDevPtr, EMPTY_CELL, size);
    hipMemset(mCellEndListDevPtr, EMPTY_CELL, size);
    
    int sharedMemSize = sizeof(int)*(mThreadsPerBlock + 1);
    compute_cell_start_end <<< mNumBlocks, mThreadsPerBlock,  sharedMemSize>>>  
        (mParticleHashListDevPtr, mCellStartListDevPtr, 
        mCellEndListDevPtr, mParameters.numParticles);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeSubParticleCellStartEndList () 
{
    int* dim = mParameters.gridDimSubParticles; 
    unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(int);

    if (mNumSubParticles == 0)
    {
        return;
    }

    hipMemset(mSubParticleCellStartIdsDevPtr, EMPTY_CELL, size);
    hipMemset(mSubParticleCellEndIdsDevPtr, EMPTY_CELL, size);
    
    int sharedMemSize = sizeof(int)*(mThreadsPerBlockSubParticle + 1);
    compute_cell_start_end 
        <<< mNumBlocksSubParticle, mThreadsPerBlockSubParticle, sharedMemSize>>>  
        (mSubParticleHashsDevPtr, mSubParticleCellStartIdsDevPtr, 
        mSubParticleCellEndIdsDevPtr, mNumSubParticles);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeDensityPressure () 
{
    compute_particle_density_pressure <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr,mParticleSimulationDataDevPtr, 
        mParticleIdsDevPtr, mCellStartListDevPtr, mCellEndListDevPtr);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeSubParticleDensityPressure () 
{
    if (mNumSubParticles == 0)
    {
        return;
    }

    compute_sub_particle_density_pressure <<< mNumBlocksSubParticle,
        mThreadsPerBlockSubParticle >>> 
        (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr, 
        mSubParticleIdsDevPtr, mSubParticleSortedIdsDevPtr, 
        mSubParticleCellStartIdsDevPtr, mSubParticleCellEndIdsDevPtr, 
        mNumParticlesSplit*8);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeAcceleration ()
{
    compute_particle_acceleration_ifsurf <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr, 
        mParticleIdsDevPtr, mCellStartListDevPtr, mCellEndListDevPtr,
        mIsSurfaceParticleDevPtr);
}
//-----------------------------------------------------------------------------
//__global__ void compute_sub_particle_acceleration
//    (float* subParticleVertexData, float* subParticleSimulationData, 
//    int* subParticleIdList, int* subParticleSortedIdList, int* cellStartList,
//    int* cellEndList, unsigned int numParticles)
void ParticleSimulation::computeSubParticleAcceleration ()
{
    if (mNumParticlesSplit == 0)
    {
        return;
    }
    
    compute_sub_particle_acceleration <<< mNumBlocksSubParticle, 
        mThreadsPerBlockSubParticle >>> 
        (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr, 
        mSubParticleIdsDevPtr, mSubParticleSortedIdsDevPtr, 
        mSubParticleCellStartIdsDevPtr, mSubParticleCellEndIdsDevPtr,
        mNumParticlesSplit*8);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::projectQuantities ()
{
    if (mNumSubParticles == 0)
    {
        return;
    }

    project_quantities 
        <<< mNumBlocksSubParticleBoundary, 
        mThreadsPerBlockSubParticleBoundary >>>
        (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr, 
        mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr, 
        mSubParticleIdsDevPtr, 8*mNumParticlesBoundary, 8*mNumParticlesSplit);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::integrate ()
{
    integrate_euler <<< mNumBlocks, mThreadsPerBlock >>>
        (mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr);
}
//-----------------------------------------------------------------------------
//__global__ void integrate_sub_particles_euler (float* subParticleVertexData, 
//    float* subParticleSimulationData, int* subParticleIds, 
//    unsigned int nSubParticles, unsigned int offset)
void ParticleSimulation::integrateSubParticles ()
{
    if (mNumSubParticles != 0)
    {
        integrate_sub_particles_euler 
            <<< mNumBlocksSubParticle, 
            mThreadsPerBlockSubParticle >>>
            (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr, 
            mSubParticleIdsDevPtr, mNumSubParticles, 0);
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::handleCollisions ()
{
    collision_handling <<< mNumBlocks, mThreadsPerBlock >>>
        (mParticleVertexDataDevPtr, mParticleSimulationDataDevPtr);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::handleSubParticleCollisions ()
{
    if (mNumSubParticles == 0)
    {
        return; 
    }
   
    collision_handling_sub_particles <<< mNumBlocksSubParticle, 
        mThreadsPerBlockSubParticle >>>
        (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr, 
        mSubParticleIdsDevPtr, mNumSubParticles, 0);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::computeParticleState ()
{
    shift_state <<< mNumBlocks, mThreadsPerBlock >>> (mParticleStatesDevPtr);
    find_split_particles <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleStatesDevPtr, mParticleIdsDevPtr,
        mCellStartListDevPtr, mCellEndListDevPtr);
    find_boundary_particles <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleVertexDataDevPtr, mParticleStatesDevPtr, mParticleIdsDevPtr,
        mCellStartListDevPtr, mCellEndListDevPtr);
}
//-----------------------------------------------------------------------------
// Collects all id's of active sub particles, particles with state "split",
// particles with state "boundary", particles with that "default" in their own
// arrays and computes the total number of each particle.
void ParticleSimulation::collect ()
{    
    CUDA_SAFE_CALL( hipMemset(_isSplitDevPtr, 0, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMemset(_isBoundaryDevPtr, 0, 
        (mParameters.numParticles + 1)*sizeof(int)) );
    CUDA_SAFE_CALL( hipMemset(_isDefaultDevPtr, 0, 
        (mParameters.numParticles + 1)*sizeof(int)) );

    check_split_boundary_default <<< mNumBlocks, mThreadsPerBlock >>> 
        (mParticleStatesDevPtr, _isSplitDevPtr, _isBoundaryDevPtr, 
        _isDefaultDevPtr);

    thrust::exclusive_scan(thrust::device_ptr<int>(_isSplitDevPtr),
        thrust::device_ptr<int>(_isSplitDevPtr + mParameters.numParticles
        + 1), thrust::device_ptr<int>(_splitPrefixSumDevPtr));
    thrust::exclusive_scan(thrust::device_ptr<int>(_isBoundaryDevPtr),
        thrust::device_ptr<int>(_isBoundaryDevPtr + mParameters.numParticles
        + 1), thrust::device_ptr<int>(_boundaryPrefixSumDevPtr));
    thrust::exclusive_scan(thrust::device_ptr<int>(_isDefaultDevPtr),
        thrust::device_ptr<int>(_isDefaultDevPtr + mParameters.numParticles
        + 1), thrust::device_ptr<int>(_defaultPrefixSumDevPtr));

    CUDA_SAFE_CALL( hipMemcpy(&mNumParticlesSplit,
        &_splitPrefixSumDevPtr[mParameters.numParticles], sizeof(int), 
        hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(&mNumParticlesBoundary,
        &_boundaryPrefixSumDevPtr[mParameters.numParticles], sizeof(int), 
        hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(&mNumParticlesDefault,
        &_defaultPrefixSumDevPtr[mParameters.numParticles], sizeof(int), 
        hipMemcpyDeviceToHost) );    

    collect_ids <<< mNumBlocks, mThreadsPerBlock >>>
        (mSubParticleIdsDevPtr, mSubParticleSortedIdsDevPtr,
        mParticleIdsSplitDevPtr, mParticleIdsBoundaryDevPtr, 
        mParticleIdsDefaultDevPtr, _isSplitDevPtr, _isBoundaryDevPtr, 
        _isDefaultDevPtr, _splitPrefixSumDevPtr, _boundaryPrefixSumDevPtr, 
        _defaultPrefixSumDevPtr, mNumParticlesSplit);

    mNumSubParticles = 8*(mNumParticlesSplit + mNumParticlesBoundary);

    // compute how many cuda blocks and how many threads a block are needed for
    // split particles, boundary particles, default particles.
    compute_particle_kernel_invocation_information(mThreadsPerBlockSplit, 
        mNumBlocksSplit, mNumParticlesSplit);
    compute_particle_kernel_invocation_information(mThreadsPerBlockBoundary, 
        mNumBlocksBoundary, mNumParticlesBoundary);
    compute_particle_kernel_invocation_information(mThreadsPerBlockDefault, 
        mNumBlocksDefault, mNumParticlesDefault);
    compute_particle_kernel_invocation_information(mThreadsPerBlockSubParticle,
        mNumBlocksSubParticle, mNumSubParticles);
    compute_particle_kernel_invocation_information
        (mThreadsPerBlockSubParticleBoundary, mNumBlocksSubParticleBoundary,
        8*mNumParticlesBoundary);
    compute_particle_kernel_invocation_information
        (mThreadsPerBlockSubParticleRegular, mNumBlocksSubParticleRegular,
        8*mNumParticlesSplit);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::initializeSubParticles () 
{
    //  initializes new sub particles if a parent particle has changed its 
    //  state from "default" to "boundary" or "split"   
    if (mNumParticlesSplit > 0)
    {
        initialize_sub_particles <<<mNumBlocksSplit, mThreadsPerBlockSplit>>> 
            (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr,
            mParticleIdsSplitDevPtr, mParticleVertexDataDevPtr, 
            mParticleSimulationDataDevPtr, mParticleStatesDevPtr, 
            mNumParticlesSplit);    
    }

    if (mNumParticlesBoundary > 0)
    {
        initialize_sub_particles <<<mNumBlocksBoundary, mThreadsPerBlockBoundary>>> 
            (mSubParticleVertexDataDevPtr, mSubParticleSimulationDataDevPtr,
            mParticleIdsBoundaryDevPtr, mParticleVertexDataDevPtr, 
            mParticleSimulationDataDevPtr, mParticleStatesDevPtr, 
            mNumParticlesBoundary);
    }
}
//-----------------------------------------------------------------------------
void ParticleSimulation::setUpSphInComplexShapes ()
{
    // set up the three dimensional textures that store the boundary 
    // information of the SPH in complex shapes paper (density contribution, 
    // distances).

    // create boundary handling data
    Wm5::Vector3f s(-0.8f, -0.6f, -0.4f); 
    Wm5::Vector3f e(0.8f, 0.6f, 0.4f);

    float h = mParameters.compactSupport;
    float particleSpacing = std::powf(mParameters.particleMass/mParameters.restDensity, 1.0f/3.0f);
    float mass = mParameters.particleMass;

    mBoundaryHandling = new SphInComplexShapes(s, e, h/4.0f, h, h, mass, 
        mParameters.restDensity, mParameters.dynamicViscosity, particleSpacing);
    
    Wm5::Box3f b(Wm5::Vector3f(0.0f, 0.0f, 0.0f), 
        Wm5::Vector3f(1.0f, 0.0f, 0.0f), Wm5::Vector3f(0.0f, 1.0f, 0.0f),
        Wm5::Vector3f(0.0f, 0.0f, 1.0f), 0.7f, 0.5f, 0.3f);
    mBoundaryHandling->SetBox(b);
    mBoundaryHandling->SaveSlicedDistanceMapToPpm("distances.ppm");
    mBoundaryHandling->SaveSlicedViscosityMapToPpm("viscosities.ppm");

    // send boundary grid information to device
    float gridOrigin[3];
    gridOrigin[0] = mBoundaryHandling->GetGridStart().X();
    gridOrigin[1] = mBoundaryHandling->GetGridStart().Y();
    gridOrigin[2] = mBoundaryHandling->GetGridStart().Z();

    unsigned int gridDimensions[3];
    gridDimensions[0] = mBoundaryHandling->GetGridDimension(0);
    gridDimensions[1] = mBoundaryHandling->GetGridDimension(1);
    gridDimensions[2] = mBoundaryHandling->GetGridDimension(2);
    float gridSpacing = mBoundaryHandling->GetGridSpacing();
    float restDistance = mBoundaryHandling->GetRestDistance();
    float gridLength[3];
    gridLength[0] = (gridDimensions[0] - 1)*gridSpacing;
    gridLength[1] = (gridDimensions[1] - 1)*gridSpacing;
    gridLength[2] = (gridDimensions[2] - 1)*gridSpacing;

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gBoundaryGridOrigin), 
        gridOrigin, 3*sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gBoundaryGridDimensions), 
       gridDimensions, 3*sizeof(unsigned int)) );    
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gBoundaryGridLength), 
       gridLength, 3*sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gBoundaryGridSpacing), 
       &gridSpacing, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(gBoundaryRestDistance), 
       &restDistance, sizeof(float)) );

    // set up 3d textures
    float* densityTexData = 
        SphInComplexShapes::CreateDensityTextureData(*mBoundaryHandling);

    set_up_3d_float_texture(&gBoundaryDensities, mBoundaryDensities, 
        densityTexData, gridDimensions);
    
    delete[] densityTexData;

    float* distanceTexData =
        SphInComplexShapes::CreateDistanceTextureData(*mBoundaryHandling);

    set_up_3d_float_texture(&gBoundaryDistances, mBoundaryDistances, 
        distanceTexData, gridDimensions);

    delete[] distanceTexData;

    float* viscosityTexData =
        SphInComplexShapes::CreateViscosityTextureData(*mBoundaryHandling);

    set_up_3d_float_texture(&gBoundaryViscosities, mBoundaryViscosities, 
        viscosityTexData, gridDimensions);

    delete[] viscosityTexData;
}
//-----------------------------------------------------------------------------
void ParticleSimulation::map () 
{
    hipGraphicsMapResources(4, mGraphicsResources);
    size_t nBytes;
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mParticleVertexDataDevPtr), &nBytes,
        mGraphicsResources[0]);
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mSubParticleVertexDataDevPtr), &nBytes,
        mGraphicsResources[1]);
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mParticleIdsDefaultDevPtr), &nBytes,
        mGraphicsResources[2]);
    hipGraphicsResourceGetMappedPointer
        (reinterpret_cast<void**>(&mSubParticleIdsDevPtr), &nBytes,
        mGraphicsResources[3]);
}

void ParticleSimulation::unmap () 
{
    hipGraphicsUnmapResources(4, mGraphicsResources);
    //cudaGLUnmapBufferObject(mParticleVertexDataVbo);
}
//-----------------------------------------------------------------------------
void ParticleSimulation::SaveInfoTable (const std::string& filename) 
{
    using namespace std;

    ofstream file;

    file.open(filename);

    int* pIdList = new int[mParameters.numParticles];
    int* pHashList = new int[mParameters.numParticles];

    int cellListSize = mParameters.gridDim[0]*mParameters.gridDim[1]*
        mParameters.gridDim[2];

    int* pCellStartList = new int[cellListSize];
    int* pCellEndList = new int[cellListSize];

    //this->map();

    hipMemcpy(pHashList, mParticleHashListDevPtr, 
        mParameters.numParticles*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pIdList, mParticleIdsDevPtr, 
        mParameters.numParticles*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pCellStartList, mCellStartListDevPtr, 
        cellListSize*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pCellEndList, mCellEndListDevPtr, 
        cellListSize*sizeof(int), hipMemcpyDeviceToHost);
    
    file << "Number of particles " << mParameters.numParticles << endl; 
    file << setw(8) << "index" << setw(12) << " id" << setw(12) << 
        " hash" << setw(12) << " start" << setw(12) << " end" << endl;

    for (unsigned int i = 0; i < cellListSize; i++) 
    {
        file << setw(8) << i;

        if(i < mParameters.numParticles)
        {
            file << setw(12) << pIdList[i];
            file << setw(12) << pHashList[i];
        } 
        else 
        {
            file << setw(12) << "";
            file << setw(12) << "";
        }

        if(pCellStartList[i] == EMPTY_CELL) 
        {
            file << setw(12) << "";
        } 
        else
        {
            file << setw(12) << pCellStartList[i];
        }
        
        if(pCellEndList[i] == EMPTY_CELL)
        {
            file << setw(12) << "" << endl;
        } 
        else
        {
            file << setw(12) << pCellEndList[i] << endl;
        }
    }

    delete[] pIdList;
    delete[] pHashList;
    delete[] pCellStartList;
    delete[] pCellEndList;

    file.close();

    //this->unmap();
}
//-----------------------------------------------------------------------------
void ParticleSimulation::SaveParticleInfo (const std::string& filename)
{
    using namespace std;

    this->map();

    ofstream file;
    
    file.open(filename);

    float* particleVertexData = 
        new float[VD_NUM_ELEMENTS*mParameters.numParticles]; 
    float* particleSimulationData = 
        new float[SD_NUM_ELEMENTS*mParameters.numParticles]; 

    // copy particle information from device to host
    hipMemcpy(particleVertexData, mParticleVertexDataDevPtr, 
		VD_NUM_ELEMENTS*mParameters.numParticles*sizeof(float), 
		hipMemcpyDeviceToHost);
    hipMemcpy(particleSimulationData, mParticleSimulationDataDevPtr, 
		SD_NUM_ELEMENTS*mParameters.numParticles*sizeof(float), 
		hipMemcpyDeviceToHost);    
    
    // set max. chars for each column of the table
    int columnWidth = 20;
    
    file << setw(columnWidth) << "Index";
    file << setw(columnWidth) << "X";
    file << setw(columnWidth) << "Y";
    file << setw(columnWidth) << "Z";
    file << setw(columnWidth) << "Density";
    file << setw(columnWidth) << "Pressure";
    file << setw(columnWidth) << "Acc X";
    file << setw(columnWidth) << "Acc Y";
    file << setw(columnWidth) << "Acc Z";
    file << endl;

    for (unsigned int i = 0; i < mParameters.numParticles; i++) 
    {
        file << setw(columnWidth) << i;
        file << setw(columnWidth) 
            << particleVertexData[VD_NUM_ELEMENTS*i + VD_POS_X];
        file << setw(columnWidth) 
            << particleVertexData[VD_NUM_ELEMENTS*i + VD_POS_Y];
        file << setw(columnWidth) 
            << particleVertexData[VD_NUM_ELEMENTS*i + VD_POS_Z];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_DENSITY];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_PRESSURE];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_ACC_X];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_ACC_Y];
        file << setw(columnWidth) 
            << particleSimulationData[SD_NUM_ELEMENTS*i + SD_ACC_Z];
        // TODO: rest of the params.
        file << endl;
    }


    delete[] particleVertexData;
    delete[] particleSimulationData;

    file.close();

    this->unmap();
}
//-----------------------------------------------------------------------------
/*
unsigned int ParticleSimulation::GetSizeMemoryGPU () const
{
    float size;
    
    size += mNumPar


}*/
//-----------------------------------------------------------------------------
//  definition of aux. functions
//-----------------------------------------------------------------------------
void create_particle_box (float sx, float sy, float sz, float d, 
    unsigned int numParticles, float** particleVD, float** particleSD,
    unsigned int* numParticlesCreated)
{
    // Creates a set of particles, that are aligned in a cube, given the starting
    // point of the box [sx, sy, sz] the length of the cube in each direction [d]
    // and the approximate amount of total particles [numParticles].
    //
    // Returns a pointer to the vertex data of the particles in [particleVD] and
    // a pointer to the simulation data of the particles in [particleSD] and the
    // actual amount of particles created

    // computed number of particles in each direction
    unsigned int num = pow(static_cast<double>(numParticles), 1.0/3.0);
    *numParticlesCreated = num*num*num;

    *particleVD = new float[*numParticlesCreated*VD_NUM_ELEMENTS];
    *particleSD = new float[*numParticlesCreated*SD_NUM_ELEMENTS];

    // check if new failed.
    if ((*particleSD) == NULL || (*particleSD) == NULL)
    {
        *numParticlesCreated = 0;
        return;
    }

    // compute spatial increment
    float dx = d/static_cast<float>(num - 1);

    // seed the particles inside the cube
    
    // set the position of each particle
    unsigned int idx;
    
    for (unsigned int k = 0; k < num; k++) 
    {
		for (unsigned int j = 0; j < num; j++) 
        {
			for (unsigned int i = 0; i < num; i++) 
            {
			    idx = VD_NUM_ELEMENTS*(num*(num*k+j)+i);
                (*particleVD)[idx + VD_POS_X] = sx + i*dx;
                (*particleVD)[idx + VD_POS_Y] = sy + j*dx;
                (*particleVD)[idx + VD_POS_Z] = sz + k*dx;
            }
		}
	}
    
    // set other particles attributes to 0.0f
    memset((*particleSD), 0, 
        sizeof(float)*SD_NUM_ELEMENTS*(*numParticlesCreated));
}
//-----------------------------------------------------------------------------
void set_simulation_domain (float xs, float ys, float zs, float xe,
    float ye, float ze, float gridSpacing, float gridSpacingSubParticles,
    SimulationParameters* parameters)
{
    // Sets the simulation domain in the [parameters], based on a starting point
    // [xs, ys, zs] an ending point [xe, ye, ze] and the distance between two
    // grid points [gridSpacing].

    parameters->gridOrigin[0] = xs;
    parameters->gridOrigin[1] = ys;
    parameters->gridOrigin[2] = zs;
    parameters->gridDim[0] = static_cast<int>((xe - xs)/gridSpacing + 0.5);
    parameters->gridDim[1] = static_cast<int>((ye - ys)/gridSpacing + 0.5);
    parameters->gridDim[2] = static_cast<int>((ze - zs)/gridSpacing + 0.5);
    parameters->gridDimSubParticles[0] = 
        static_cast<int>((xe - xs)/gridSpacingSubParticles + 0.5);
    parameters->gridDimSubParticles[1] = 
        static_cast<int>((ye - ys)/gridSpacingSubParticles + 0.5);
    parameters->gridDimSubParticles[2] = 
        static_cast<int>((ze - zs)/gridSpacingSubParticles + 0.5);
    parameters->gridSpacing = gridSpacing;
    parameters->gridSpacingSubParticles = gridSpacingSubParticles;
}
//-----------------------------------------------------------------------------
void compute_particle_kernel_invocation_information
    (unsigned int& nThreadsBlock, unsigned int& nBlocks, 
    unsigned int numParticles)
{
    if (numParticles == 0)
    {
        nThreadsBlock = 0;
        nBlocks = 0;
        return;
    }

    nThreadsBlock = numParticles > 256 ? 256 : numParticles;
    nBlocks = numParticles % nThreadsBlock == 0 ? numParticles/nThreadsBlock : 
        numParticles/nThreadsBlock + 1;
}
//-----------------------------------------------------------------------------
void set_up_3d_float_texture (struct textureReference* texRef, 
    hipArray* arr, float* data, unsigned int dim[3])
{
    // debug! set all arr vals to 0.5f
    /*for (unsigned int i = 0; i < dim[0]*dim[1]*dim[2]; i++)
    {
        if (i % 2)
        {
            data[i] = 1.0f;
        }
        else
        {
            data[i] = 0.5f;
        }
    }*/


    // set allocation parmeters
    hipChannelFormatDesc descf = hipCreateChannelDesc(32, 0, 0, 0,
		hipChannelFormatKindFloat);
    hipExtent extent;
    extent.width = dim[0];
    extent.height = dim[1];
    extent.depth = dim[2];

    // alloc cuda array
    CUDA_SAFE_CALL( hipMalloc3DArray(&arr, &descf, extent) );
    
    // set copy parameters
    hipMemcpy3DParms copyParams;
    memset(&copyParams, 0, sizeof(hipMemcpy3DParms));
    
    copyParams.srcPtr = make_hipPitchedPtr((void *)data, 
        extent.width*sizeof(float), extent.width, extent.height);
    
    copyParams.dstArray = arr;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;

    // transfer 3d data to cuda array
    CUDA_SAFE_CALL( hipMemcpy3D(&copyParams) );


    // set texture parameters
    texRef->normalized = true;                      
    texRef->filterMode = hipFilterModeLinear;      
    texRef->addressMode[0] = hipAddressModeClamp;   
    texRef->addressMode[1] = hipAddressModeClamp;
    texRef->addressMode[2] = hipAddressModeClamp;

    // bind array to global texture 
    hipBindTextureToArray(texRef, arr, &descf);
}
